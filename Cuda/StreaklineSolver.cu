#include "StreaklineSolver.h"
#include "CudaHelperFunctions.h"




bool StreaklineSolver::release()
{
	this->volume_IO.release();
	hipFree(this->d_Particles);

	this->volumeTexture_0.release();
	this->volumeTexture_1.release();

	return true;
}

__host__ bool StreaklineSolver::initializeRealtime(SolverOptions * p_solverOptions, FieldOptions * p_fieldOptions)
{

	this->solverOptions = p_solverOptions;
	this->fieldOptions = p_fieldOptions;
	this->InitializeCUDA();
	this->volume_IO.Initialize(fieldOptions);
	this->initializeParticles();



	int blockDim = 256;
	int thread = (this->solverOptions->lines_count / blockDim) + 1;

	// set the position of the vertex buffer to the intial position of the particle
	InitializeVertexBufferStreaklines << <blockDim, thread >> >
		(this->d_Particles,
			*solverOptions,
			reinterpret_cast<Vertex*>(this->p_VertexBuffer)
			);
	
	return true;
}

__host__ bool StreaklineSolver::solve()
{
	//At least two timesteps is needed
	int timeSteps = solverOptions->lastIdx - solverOptions->currentIdx;

	// Initialize Volume IO (Save file path and file names)
	this->volume_IO.Initialize(this->fieldOptions);

	// Initialize Particles and upload it to GPU
	this->initializeParticles();

	// Number of threads based on the number of lines
	dim3 thread = { maxBlockDim,maxBlockDim,1 };
	int blocks = BLOCK_THREAD(this->solverOptions->lines_count);

	solverOptions->lineLength = timeSteps;
	bool odd = false;

	// set the position of the vertex buffer to the intial position of the particle
	InitializeVertexBufferStreaklines << <blocks, thread >> >
		(	this->d_Particles,
			*solverOptions,
			reinterpret_cast<Vertex*>(this->p_VertexBuffer)
			);

	// we go through each time step and solve RK4 for even time steps the first texture is updated,
	// while the second texture is updated for odd time steps
	for (int step = 0; step < timeSteps; step++)
	{
		// First Step
		if (step == 0)
		{

			// Read current volume
			this->volume_IO.readVolume(solverOptions->currentIdx);
			// Return a pointer to volume
			this->h_VelocityField = this->volume_IO.getField_float();
			// set the pointer to the volume texture
			this->volumeTexture_0.setField(h_VelocityField);
			// initialize the volume texture
			this->volumeTexture_0.initialize(Array2Int3(solverOptions->gridSize), true, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
			// release host memory
			volume_IO.release();



			// same procedure for the second field
			this->volume_IO.readVolume(solverOptions->currentIdx + 1);
			this->h_VelocityField = this->volume_IO.getField_float();
			this->volumeTexture_1.setField(h_VelocityField);
			this->volumeTexture_1.initialize(Array2Int3(solverOptions->gridSize), true, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);

			volume_IO.release();

		}

		else if (step % 2 == 0) // => EVEN
		{
			this->volume_IO.readVolume(solverOptions->currentIdx + step + 1);
			this->h_VelocityField = this->volume_IO.getField_float();

			this->volumeTexture_1.release();
			this->volumeTexture_1.setField(h_VelocityField);
			this->volumeTexture_1.initialize(Array2Int3(solverOptions->gridSize), true, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);

			volume_IO.release();


			odd = false;
		}

		else if (step % 2 != 0) // => ODD
		{

			this->volume_IO.readVolume(solverOptions->currentIdx + step + 1);
			this->h_VelocityField = this->volume_IO.getField_float();

			this->volumeTexture_0.release();
			this->volumeTexture_0.setField(h_VelocityField);
			this->volumeTexture_0.initialize(Array2Int3(solverOptions->gridSize), true, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);

			volume_IO.release();

			odd = true;

		}

		TracingStreak << <blocks, thread >> >
			(
				volumeTexture_0.getTexture(),
				volumeTexture_1.getTexture(),
				*solverOptions,
				reinterpret_cast<Vertex*>(this->p_VertexBuffer),
				odd,
				step
				);



	}

	// Bring the position to the middle
	AddOffsetVertexBufferStreaklines << <blocks, thread >> >(*solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer));
	this->release();
	return true;
}


__host__ bool StreaklineSolver::solveRealtime(int & streakCounter)
{
	dim3 thread = { maxBlockDim,maxBlockDim,1 };
	int blocks = BLOCK_THREAD(this->solverOptions->lines_count);

	bool odd = false;


	// First Step

	switch (fieldOptions->isCompressed)
	{

	case true: // Compressed Data
	{
		if (streakCounter == 0)
		{
			loadTextureCompressed( volumeTexture_0, solverOptions->firstIdx);
			loadTextureCompressed( volumeTexture_1, solverOptions->firstIdx + 1);
		}
		else if (streakCounter % 2 == 0) // => EVEN
		{
			this->volumeTexture_1.release();
			loadTextureCompressed( volumeTexture_1, solverOptions->firstIdx + streakCounter + 1);
			odd = false;
		}
		else if (streakCounter % 2 != 0) // => ODD
		{
			this->volumeTexture_0.release();
			loadTextureCompressed( volumeTexture_0, solverOptions->firstIdx + streakCounter + 1);
			odd = true;

		}

		break;
	}

	case false: // Uncompressed Data
	{
		if (streakCounter == 0)
		{
			loadTexture( volumeTexture_0, solverOptions->firstIdx);
			loadTexture( volumeTexture_1, solverOptions->firstIdx + 1);
		}
		else if (streakCounter % 2 == 0) // => EVEN
		{
			this->volumeTexture_1.release();
			loadTexture( volumeTexture_1, solverOptions->firstIdx + streakCounter + 1);
			odd = false;

		}
		else if (streakCounter % 2 != 0) // => ODD
		{
			this->volumeTexture_0.release();
			loadTexture( volumeTexture_0, solverOptions->firstIdx + streakCounter + 1);
			odd = true;
		}
		break;
	}

	}





	if (streakCounter == solverOptions->lineLength - 1)
	{

		resetRealtime();
		streakCounter = 0;

	}
	else
	{
		
	
	TracingStreak << <blocks, thread >> > (volumeTexture_0.getTexture(), volumeTexture_1.getTexture(), *solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer), odd, streakCounter);
	std::printf("\n\n");
	streakCounter++;

	}
	solverOptions->counter = streakCounter;

	return true;
}


__host__ bool StreaklineSolver::resetRealtime()
{

	this->release();

	return true;
}
