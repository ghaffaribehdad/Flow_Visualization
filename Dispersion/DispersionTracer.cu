#include "hip/hip_runtime.h"
#include "DispersionTracer.h"
#include "DispersionHelper.h"
#include "..//ErrorLogger/ErrorLogger.h"
#include "..//Raycaster/IsosurfaceHelperFunctions.h"
#include <hip/hip_runtime.h>
#include "..//Raycaster/Raycasting_Helper.h"
#include "..//Options/DispresionOptions.h"
#include "DispersionHelper.h"

//explicit instantiation

bool HeightfieldGenerator::retrace()
{
	this->a_HeightSurface_Primary.release();
	this->a_HeightSurface_Primary_Extra.release();

	hipDestroyTextureObject(this->volumeTexture3D_height.getTexture());
	hipDestroyTextureObject(this->volumeTexture3D_height_extra.getTexture());

	hipFree(d_particle);

	if (!this->InitializeParticles())
		return false;

	// Initialize Height Field as an empty cuda array 3D
	if (!this->singleSurfaceInitialization())
		return false;

	return true;
}

bool HeightfieldGenerator::initialize
(
	hipTextureAddressMode addressMode_X ,
	hipTextureAddressMode addressMode_Y ,
	hipTextureAddressMode addressMode_Z 
)
{

	if (!this->initializeRaycastingTexture())				// initilize texture (the texture we need to write to)
		return false;


	if (!this->initializeBoundingBox())		// initialize the bounding box ( copy data to the constant memory of GPU about Bounding Box)
		return false;


	// set the number of rays = number of pixels
	this->rays = (*this->width) * (*this->height);

	if (!this->InitializeParticles())
		return false;


	singleSurfaceInitialization();
	
	return true;
}

void HeightfieldGenerator::setResources(Camera* _camera,
	int* _width,
	int* _height,
	SolverOptions* _solverOption,
	RaycastingOptions* _raycastingOptions,
	RenderingOptions* _renderingOptions,
	ID3D11Device* _device,
	IDXGIAdapter* _pAdapter,
	ID3D11DeviceContext* _deviceContext,
	DispersionOptions* _dispersionOptions)
{
	Raycasting::setResources(_camera, _width,_height,_solverOption,_raycastingOptions,_renderingOptions,_device,_pAdapter,_deviceContext);
		this->dispersionOptions		= _dispersionOptions;
}


__host__ bool HeightfieldGenerator::InitializeParticles()
{
	this->n_particles = dispersionOptions->gridSize_2D[0] * dispersionOptions->gridSize_2D[1];
	this->h_particle = new Particle[n_particles];
	seedParticle_tiltedPlane
	(
		h_particle,
		Array2Float3(solverOptions->gridDiameter),
		ARRAYTOINT2(dispersionOptions->gridSize_2D),
		dispersionOptions->seedWallNormalDist,
		dispersionOptions->tilt_deg
	);

	size_t Particles_byte = sizeof(Particle) * n_particles;

	gpuErrchk(hipMalloc((void**)& this->d_particle, Particles_byte));
	gpuErrchk(hipMemcpy(this->d_particle, this->h_particle, Particles_byte, hipMemcpyHostToDevice));

	delete[] h_particle;

	return true;
}



__host__ bool HeightfieldGenerator::InitializeHeightArray3D_Single(int x, int y, int z)
{
	// Set dimensions and initialize height field as a 3D CUDA Array
	this->a_HeightSurface_Primary.setDimension(x, y, z);


	this->a_HeightSurface_Primary_Extra.setDimension(x, y, z);

	// initialize the 3D array
	if (!a_HeightSurface_Primary.initialize())
		return false;
	if (!a_HeightSurface_Primary_Extra.initialize())
		return false;

	return true;
}




__host__ bool HeightfieldGenerator::InitializeHeightArray3D_Single(int3 gridSize)
{
	// Set dimensions and initialize height field as a 3D CUDA Array
	this->a_HeightSurface_Primary.setDimension(gridSize.x, gridSize.y, gridSize.z);


	this->a_HeightSurface_Primary_Extra.setDimension(gridSize.x, gridSize.y, gridSize.z);

	// initialize the 3D array
	if (!a_HeightSurface_Primary.initialize())
		return false;
	if (!a_HeightSurface_Primary_Extra.initialize())
		return false;

	return true;
}


__host__ bool HeightfieldGenerator::InitializeHeightSurface3D_Single()
{
	// Assign the hightArray to the hightSurface and initialize the surface
	this->s_HeightSurface_Primary.setInputArray(a_HeightSurface_Primary.getArrayRef());
	if (!this->s_HeightSurface_Primary.initializeSurface())
		return false;

	this->s_HeightSurface_Primary_Extra.setInputArray(a_HeightSurface_Primary_Extra.getArrayRef());
	if (!this->s_HeightSurface_Primary_Extra.initializeSurface())
		return false;

	return true;
}




// Release resources 
bool HeightfieldGenerator::release()
{
	Raycasting::release();
	hipDestroyTextureObject(this->volumeTexture3D_height.getTexture());
	this->a_HeightSurface_Primary.release();

	return true;
}

void HeightfieldGenerator::trace3D_path_Single()
{
	// Calculates the block and grid sizes
	unsigned int blocks;
	dim3 thread = { maxBlockDim,maxBlockDim,1 };
	blocks = BLOCK_THREAD(n_particles);

	RK4STEP RK4Step = RK4STEP::EVEN;
	
	for (int i = 0; i < solverOptions->lastIdx - solverOptions->firstIdx ; i++)
	{
		if (i == 0) // initial time step
		{
			
			// Load i 'dx field in volume_IO into field
			volume_IO.readVolume(i + solverOptions->currentIdx);
			// Copy and initialize velocityfield texture
			t_velocityField_0.setField(volume_IO.getField_float());
			t_velocityField_0.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
			// Release the velocityfield from host (volume_IO)
			volume_IO.release();



			// Same procedure for the second texture
			volume_IO.readVolume(i + solverOptions->currentIdx + 1);
			
			t_velocityField_1.setField(volume_IO.getField_float());
			t_velocityField_1.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
			// Release the velocityfield from host (volume_IO)
			volume_IO.release();

		}
		else
		{
			// Even integration steps
			if (i % 2 == 0)
			{
				
				volume_IO.readVolume(solverOptions->currentIdx + i + 1);
				t_velocityField_1.release();
				t_velocityField_1.setField(volume_IO.getField_float());
				t_velocityField_1.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
				volume_IO.release();

				RK4Step = RK4STEP::EVEN;
			}
			else
			{
				volume_IO.readVolume(solverOptions->currentIdx + i +1);
				t_velocityField_0.release();
				t_velocityField_0.setField(volume_IO.getField_float());
				t_velocityField_0.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
				volume_IO.release();

				RK4Step = RK4STEP::ODD;

			}

		}

		traceDispersion3D_path << < blocks, thread >> >
			(
				d_particle,
				s_HeightSurface_Primary.getSurfaceObject(),
				s_HeightSurface_Primary_Extra.getSurfaceObject(),
				this->t_velocityField_0.getTexture(),
				this->t_velocityField_1.getTexture(),
				*solverOptions,
				*dispersionOptions,
				RK4Step,
				i
			);
	}



	// Calculates the gradients and store it in the cuda surface
	hipFree(d_particle);
}








__host__ void HeightfieldGenerator::rendering()
{
	this->deviceContext->PSSetSamplers(0, 1, this->samplerState.GetAddressOf());
	//this->deviceContext->OMSetBlendState(this->blendState.Get(), NULL, 0xFFFFFFFF);

	this->deviceContext->ClearRenderTargetView(this->renderTargetView.Get(), renderingOptions->bgColor);// Clear the target view

	// Calculates the block and grid sizes
	unsigned int blocks;
	dim3 thread = { maxBlockDim,maxBlockDim,1 };
	blocks = static_cast<unsigned int>((this->rays % (thread.x * thread.y) == 0 ? rays / (thread.x * thread.y) : rays / (thread.x * thread.y) + 1));
 
	

	// Depending on the Rendering mode choose the terrain Rendering function
	if (dispersionOptions->renderingMode == dispersionOptionsMode::HeightfieldRenderingMode::SINGLE_SURFACE)
	{
		CudaTerrainRenderer_Marching_extra<< < blocks, thread >> >
			(
				this->raycastingSurface.getSurfaceObject(),
				this->volumeTexture3D_height.getTexture(),
				this->volumeTexture3D_height_extra.getTexture(),
				int(this->rays),
				this->raycastingOptions->samplingRate_0,
				this->raycastingOptions->tolerance_0,
				*dispersionOptions,
				solverOptions->lastIdx - solverOptions->firstIdx
				);
	}




}


bool HeightfieldGenerator::updateScene()
{
	if (!this->initializeRaycastingInteroperability())	// Create interoperability while we need to release it at the end of rendering
		return false;

	if (!this->initializeCudaSurface())					// reinitilize cudaSurface	
		return false;

	if (!this->initializeBoundingBox())					//updates constant memory
		return false;


	this->rendering();


	if (!this->raycastingSurface.destroySurface())
		return false;

	this->interoperatibility.release();


	return true;
}



void HeightfieldGenerator::gradient3D_Single()
{

	// Calculates the block and grid sizes
	unsigned int blocks;
	dim3 thread = { maxBlockDim,maxBlockDim,1 };
	blocks = static_cast<unsigned int>((this->n_particles % (thread.x * thread.y) == 0 ?
		n_particles / (thread.x * thread.y) : n_particles / (thread.x * thread.y) + 1));

	heightFieldGradient3D<FetchTextureSurface::Channel_X> << < blocks, thread >> >
		(
			s_HeightSurface_Primary.getSurfaceObject(),
			*dispersionOptions,
			*solverOptions
		);


}


bool HeightfieldGenerator::singleSurfaceInitialization()
{
	// initialize volume Input Output
	volume_IO.Initialize(this->solverOptions);


	// Initialize Height Field as an empty cuda array 3D
	if (!this->InitializeHeightArray3D_Single
	(
		dispersionOptions->gridSize_2D[0],
		dispersionOptions->gridSize_2D[1],
		solverOptions->lastIdx - solverOptions->firstIdx
	))
		return false;



	// Bind the array of heights to the cuda surface
	if (!this->InitializeHeightSurface3D_Single())
		return false;


	// Trace particle and store their heights on the Height Surface
	this->trace3D_path_Single();


	// Store gradient and height on the surface
	this->gradient3D_Single();


	this->s_HeightSurface_Primary.destroySurface();
	this->s_HeightSurface_Primary_Extra.destroySurface();

	volumeTexture3D_height.setArray(a_HeightSurface_Primary.getArrayRef());
	volumeTexture3D_height.initialize_array(false,hipAddressModeClamp, hipAddressModeClamp, hipAddressModeClamp);

	volumeTexture3D_height_extra.setArray(a_HeightSurface_Primary_Extra.getArrayRef());
	volumeTexture3D_height_extra.initialize_array(false, hipAddressModeClamp, hipAddressModeClamp, hipAddressModeClamp);

	return true;
}



bool HeightfieldGenerator::initializeShaders()
{

	if (this->vertexBuffer.Get() == nullptr)
	{
		std::wstring shaderfolder;
#pragma region DetermineShaderPath
		if (IsDebuggerPresent() == TRUE)
		{
#ifdef _DEBUG //Debug Mode
#ifdef _WIN64 //x64
			shaderfolder = L"x64\\Debug\\";
#else //x86
			shaderfolder = L"Debug\\"
#endif // DEBUG
#else //Release mode
#ifdef _WIN64 //x64
			shaderfolder = L"x64\\Release\\";
#else  //x86
			shaderfolder = L"Release\\"
#endif // Release
#endif // _DEBUG or Release mode
		}

		D3D11_INPUT_ELEMENT_DESC layout[] =
		{
			{
				"POSITION",
				0,
				DXGI_FORMAT::DXGI_FORMAT_R32G32B32_FLOAT,
				0,
				D3D11_APPEND_ALIGNED_ELEMENT,
				D3D11_INPUT_CLASSIFICATION::D3D11_INPUT_PER_VERTEX_DATA,
				0
			},

			{
				"TEXCOORD",
				0,
				DXGI_FORMAT::DXGI_FORMAT_R32G32_FLOAT,
				0,
				D3D11_APPEND_ALIGNED_ELEMENT,
				D3D11_INPUT_CLASSIFICATION::D3D11_INPUT_PER_VERTEX_DATA,
				0
			}
		};

		UINT numElements = ARRAYSIZE(layout);

		if (!vertexshader.Initialize(this->device, shaderfolder + L"vertexshaderTexture.cso", layout, numElements))
			return false;

		// Depending on the Rendering mode initialize single or double surface
		if (dispersionOptions->renderingMode == dispersionOptionsMode::HeightfieldRenderingMode::SINGLE_SURFACE)
		{
			if (!pixelshader.Initialize(this->device, shaderfolder + L"pixelshaderTextureSampler.cso"))
				return false;
		}
		else
		{
			if (!pixelshader.Initialize(this->device, shaderfolder + L"pixelshaderTextureSampler_Double.cso"))
				return false;
		}
		
	}


	return true;
}