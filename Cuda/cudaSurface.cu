#include "cudaSurface.cuh"



bool CudaSurface::initializeSurface()
{

	// Allocate CUDA arrays in device memory
	hipChannelFormatDesc channelDesc =
		hipCreateChannelDesc(32, 32, 32, 32,
			hipChannelFormatKindFloat);
	

	hipMallocArray(&this->cuInputArray, &channelDesc, this->width, this->height,
		hipArraySurfaceLoadStore);



	// Specify surface
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;

	// Create the surface objects
	resDesc.res.array.array = this->cuInputArray;
	hipCreateSurfaceObject(&surfaceObject, &resDesc);

	return true;
}


bool CudaSurface::destroySurface()
{
	// Destroy surface objects
	hipDestroySurfaceObject(surfaceObject);

	// Free device memory
	hipFreeArray(cuInputArray);

	return true;
}

hipSurfaceObject_t CudaSurface::getSurfaceObject()
{
	return surfaceObject;
}