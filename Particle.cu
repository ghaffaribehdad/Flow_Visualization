#include "hip/hip_runtime.h"
#include "Particle.cuh"

// Explicit instantions
template class Particle<float>;
template class Particle<double>;

template <typename T>
__device__  void Particle<T>::updateVelocity(const float3& gridDiameter, const int3& gridSize, hipTextureObject_t t_VelocityField)
{
	if (!outOfScope)
	{
		float3 relativePos = findRelative(gridDiameter);
		float4 velocity4D = tex3D<float4>(t_VelocityField, relativePos.z, relativePos.y, relativePos.x);
		float3 velocity = { velocity4D.x,velocity4D.y,velocity4D.z };
		this->setVelocity(velocity);
	}

}


template <typename T>
__device__ __host__ float3 Particle<T>::findRelative(const float3& gridDiameter)
{
	float3 relative_position = {
		(static_cast<float>(this->m_position.x)) / (gridDiameter.x),
		(static_cast<float>(this->m_position.y)) / (gridDiameter.y),
		(static_cast<float>(this->m_position.z)) / (gridDiameter.z)
	};
	return relative_position;
}

template <typename T>
__host__ void  Particle<T>::seedParticle(const float* gridDiameter, const float* seedBox, const float* seedBoxPos)
{
	this->m_position.x = +gridDiameter[0] /	2.0f - seedBox[0] / 2.0f + seedBoxPos[0] + static_cast <float> (rand()) / static_cast <float> (RAND_MAX / seedBox[0]);
	this->m_position.y = +gridDiameter[1] / 2.0f - seedBox[1] / 2.0f + seedBoxPos[1] + static_cast <float> (rand()) / static_cast <float> (RAND_MAX / seedBox[1]);
	this->m_position.z = +gridDiameter[2] / 2.0f - seedBox[2] / 2.0f + seedBoxPos[2] + static_cast <float> (rand()) / static_cast <float> (RAND_MAX / seedBox[2]);

}

template <typename T>
__device__ void Particle<T>::checkPosition(const float3& gridDiameter)
{

	if (m_position.x >= gridDiameter.x)
	{
		m_position.x = gridDiameter.x;
		this->outOfScope = true;
	}
	else if (m_position.y >= gridDiameter.y)
	{
		m_position.y = gridDiameter.y;
		this->outOfScope = true;
	}
	else if (m_position.z >= gridDiameter.z)
	{
		m_position.z = gridDiameter.z;
		this->outOfScope = true;
	}

}

template <typename T>
__device__ void Particle<T>::updatePosition(const float dt)
{
	if (!outOfScope)
	{
		this->m_position.x += dt * (this->m_velocity.x);
		this->m_position.y += dt * (this->m_velocity.y);
		this->m_position.z += dt * (this->m_velocity.z);
	}

}

