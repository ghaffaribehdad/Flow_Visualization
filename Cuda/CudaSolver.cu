#include "hip/hip_runtime.h"
#include "CudaSolver.h"
#include "..//Cuda/CudaHelperFunctions.h"
#include "..//ErrorLogger/ErrorLogger.h"
#include "../Particle/ParticleHelperFunctions.h"
#include "..//VolumeTexture/VolumeTexture.h"
#include "../Timer/Timer.h"
CUDASolver::CUDASolver()
{
	//std::printf("A solver is created!\n");
}

// Initilize the solver
bool CUDASolver::Initialize(SolverOptions * _solverOptions)
{
	this->solverOptions = _solverOptions;
	this->InitializeCUDA();
	this->volume_IO.Initialize(_solverOptions);
	
	return true;
}

bool CUDASolver::Reinitialize()
{
	this->InitializeCUDA();

	return true;
}

void CUDASolver::releaseVolumeIO()
{
	this->volume_IO.release();
}


bool SeedFiled(SeedingPattern, DirectX::XMFLOAT3 dimenions, DirectX::XMFLOAT3 seedbox)
{
	return true;
}


bool CUDASolver::FinalizeCUDA()
{
	gpuErrchk(hipGraphicsUnmapResources(1, &this->cudaGraphics));

	gpuErrchk(hipGraphicsUnregisterResource(this->cudaGraphics));
	
	return true;
}

bool CUDASolver::InitializeCUDA()
{

	// Register Vertex Buffer to map it
	gpuErrchk(cudaGraphicsD3D11RegisterResource(
		&this->cudaGraphics,
		this->solverOptions->p_vertexBuffer,
		hipGraphicsRegisterFlagsNone));

	// Map Vertex Buffer
	gpuErrchk(hipGraphicsMapResources(
		1,
		&this->cudaGraphics
		));

	// Get Mapped pointer
	size_t size = static_cast<size_t>(solverOptions->lines_count)* static_cast<size_t>(solverOptions->lineLength)*sizeof(Vertex);

	gpuErrchk(hipGraphicsResourceGetMappedPointer(
		&p_VertexBuffer,
		&size,
		this->cudaGraphics
	));

	return true;
}






void CUDASolver::InitializeParticles(SeedingPattern seedingPattern)
{


	// Create an array of particles
	this->h_Particles = new Particle[solverOptions->lines_count];

	switch (seedingPattern)
	{


		case SeedingPattern::SEED_RANDOM:
		{
			// Seed Particles Randomly according to the grid diameters
			for (int i = 0; i < solverOptions->lines_count; i++)
			{
				seedParticleRandom(h_Particles, solverOptions);
			}
			break;
		}

		case SeedingPattern::SEED_GRIDPOINTS:
		{
			// Create an array of particles

			seedParticleGridPoints(this->h_Particles, solverOptions);
			break;
		}

		case SeedingPattern::SEED_TILTED_PLANE:
		{
			float3 gridDiamter = Array2Float3(solverOptions->gridDiameter);
			seedParticle_tiltedPlane
			(
				this->h_Particles,
				gridDiamter,
				make_int2(solverOptions->gridSize_2D[0], solverOptions->gridSize_2D[1]),
				solverOptions->seedWallNormalDist,
				solverOptions->tilt_deg
			);
		}
		case SeedingPattern::SEED_FILE:
		{
			break;
		}


	}

	size_t Particles_byte = sizeof(Particle) * solverOptions->lines_count;

	// Upload Velocity Filled to GPU 

	gpuErrchk(hipMalloc((void**) &this->d_Particles, Particles_byte));

	gpuErrchk(hipMemcpy(this->d_Particles, this->h_Particles, Particles_byte, hipMemcpyHostToDevice));

	delete[] this->h_Particles;
}

void CUDASolver::loadTexture
(
	SolverOptions * solverOptions,
	VolumeTexture3D & volumeTexture,
	const int & idx,
	hipTextureAddressMode addressModeX ,
	hipTextureAddressMode addressModeY ,
	hipTextureAddressMode addressModeZ 
)
{
	// Read current volume
	this->volume_IO.readVolume(idx);
	// Return a pointer to volume
	float * h_VelocityField = this->volume_IO.getField_float();
	// set the pointer to the volume texture
	volumeTexture.setField(h_VelocityField);
	// initialize the volume texture
	volumeTexture.initialize(Array2Int3(solverOptions->gridSize), true, addressModeX, addressModeY, addressModeZ);
	// release host memory
	volume_IO.release();
}


void CUDASolver::loadTextureCompressed
(
	SolverOptions * solverOptions,
	VolumeTexture3D & volumeTexture,
	const int & idx,
	hipTextureAddressMode addressModeX,
	hipTextureAddressMode addressModeY,
	hipTextureAddressMode addressModeZ
)
{

	Timer timer;

	// Read current volume
	this->volume_IO.readVolume(idx, solverOptions);
	// Return a pointer to volume
	float * h_VelocityField = this->volume_IO.getField_float_GPU();
	// set the pointer to the volume texture
	volumeTexture.setField(h_VelocityField);
	// initialize the volume texture
	TIMELAPSE(volumeTexture.initialize_devicePointer(Array2Int3(solverOptions->gridSize), true, addressModeX, addressModeY, addressModeZ),"Initialize Texture including DDCopy");
	// release host memory
	//volume_IO.release();
	hipFree(h_VelocityField);
}