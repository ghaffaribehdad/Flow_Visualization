#include "hip/hip_runtime.h"

#include "VolumeTexture.h"
#include <string>






hipTextureObject_t VolumeTexture::initialize
(
	hipTextureAddressMode addressMode_x ,
	hipTextureAddressMode addressMode_y ,
	hipTextureAddressMode addressMode_z
)
{

	hipExtent extent = make_hipExtent(this->solverOptions->gridSize[0], this->solverOptions->gridSize[1], this->solverOptions->gridSize[2]);

	// Allocate 3D Array
	hipChannelFormatDesc channelFormatDesc = hipCreateChannelDesc<float4>();
	hipMalloc3DArray(&this->cuArray_velocity , &channelFormatDesc, extent);



	// set copy parameters to copy from velocity field to array
	hipMemcpy3DParms cpyParams = { 0 };

	cpyParams.srcPtr = make_hipPitchedPtr((void*)this->h_field,extent.width * sizeof(float4),extent.width, extent.height);
	cpyParams.dstArray = this->cuArray_velocity;
	cpyParams.kind = hipMemcpyHostToDevice;
	cpyParams.extent = extent;
	
	// Copy velocities to 3D Array
	gpuErrchk(hipMemcpy3D(&cpyParams));
	// might need sync before release the host memory


	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description
	texDesc.normalizedCoords = true;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.addressMode[0] = addressMode_x;
	texDesc.addressMode[1] = addressMode_y;
	texDesc.addressMode[2] = addressMode_z;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return t_field;

}
void VolumeTexture::release()
{
	hipFreeArray(this->cuArray_velocity);
	hipDestroyTextureObject(this->t_field);
}



