#include "hip/hip_runtime.h"
#include "StreamlineSolver.h"
#include "hip/hip_vector_types.h"
#include "..//Cuda/CudaHelperFunctions.h"
#include ""
#include "..//VolumeIO/BinaryWriter.h"




__host__ void StreamlineSolver::release()
{
	hipFree(this->d_Particles);
	//hipFree(this->d_VelocityField);
	
}

__host__ bool StreamlineSolver::solve()
{
	if (solverOptions->fileChanged)
	{
		// Read Dataset
		this->volume_IO.Initialize(this->solverOptions);
		this->volume_IO.readVolume(this->solverOptions->currentIdx);

		this->h_VelocityField = this->volume_IO.getField_float();

		// Release the texture
		if (solverOptions->fileLoaded)
		{
			this->volumeTexture.release();
		}
		


		// Copy data to the texture memory
		this->volumeTexture.setField(h_VelocityField);
		this->volumeTexture.initialize(Array2Int3(solverOptions->gridSize));


		// Release it from Host
		volume_IO.release();
		solverOptions->fileChanged = false;
		solverOptions->fileLoaded = true;
	}

	this->InitializeParticles(static_cast<SeedingPattern>( this->solverOptions->seedingPattern));

	int blockDim = 1024;
	int thread = (this->solverOptions->lines_count / blockDim)+1;

	
	TracingStream << <blockDim , thread >> > (this->d_Particles, volumeTexture.getTexture(), *this->solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer));

	this->release();

	return true;
}


//__host__ bool StreamlineSolver::solveAndWrite()
//{
//	// Read Dataset
//	//this->volume_IO.Initialize(this->solverOptions);
//	//this->volume_IO.readVolume(this->solverOptions->currentIdx);
//
//	//this->h_VelocityField = this->volume_IO.flushBuffer_float();
//
//	// Copy data to the texture memory
//	//this->volumeTexture.setField(h_VelocityField);
//	//this->volumeTexture.setSolverOptions(this->solverOptions);
//	//this->volumeTexture.initialize();
//
//	this->measureFieldGeneration();
//	// Release it from Host
//	//volume_IO.release();
//
//	float4* d_vertexBuffer;
//	float4* h_vertexBuffer = new float4[solverOptions->lines_count * solverOptions->lineLength];
//	hipMalloc(&d_vertexBuffer, solverOptions->lines_count * solverOptions->lineLength * sizeof(float4));
//
//
//	this->InitializeParticles(static_cast<SeedingPattern>(this->solverOptions->seedingPattern));
//
//	int blockDim = 1024;
//	int thread = (this->solverOptions->lines_count / blockDim) + 1;
//
//	InitializeVorticityTexture();
//	
//	TracingStream << <blockDim, thread >> > (this->d_Particles, volumeTexture.getTexture(),*this->solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer), d_vertexBuffer);
//
//	this->release();
//
//	hipMemcpy(h_vertexBuffer, d_vertexBuffer, solverOptions->lines_count * solverOptions->lineLength * sizeof(float4), hipMemcpyDeviceToHost);
//
//	BinaryWriter binaryWriter;
//	std::string fileName = std::string(this->solverOptions->fileName_out);
//	fileName += "_";
//	fileName += std::to_string(solverOptions->counter);
//
//	binaryWriter.setFileName(fileName);
//	binaryWriter.setFilePath(this->solverOptions->filePath_out);
//	binaryWriter.setBufferSize(solverOptions->lines_count * solverOptions->lineLength * sizeof(float4));
//	binaryWriter.setBuffer(reinterpret_cast<char*>(h_vertexBuffer));
//	binaryWriter.write();
//	
//	hipFree(d_vertexBuffer);
//	delete[] h_vertexBuffer;
//	hipDestroyTextureObject(t_measure);
//	this->a_Measure.release();
//	return true;
//}


//__host__ void StreamlineSolver::measureFieldGeneration()
//{
//	// Calculates the block and grid sizes
//	unsigned int blocks;
//	dim3 thread = { 16,16,1 };
//	int gridpoints = solverOptions->gridSize[0];
//	blocks = static_cast<unsigned int>((gridpoints % (thread.x * thread.y) == 0 ?
//		gridpoints / (thread.x * thread.y) : gridpoints / (thread.x * thread.y) + 1));
//
//
//	this->a_Measure.initialize
//	(
//		solverOptions->gridSize[0],
//		solverOptions->gridSize[1],
//		solverOptions->gridSize[2]
//	);
//
//	// initialize the CUDA surface 
//
//	this->s_Measure.setInputArray(this->a_Measure.getArrayRef());
//	this->s_Measure.initializeSurface();
//
//
//	// initialize velocity volume
//	// Read Dataset
//	this->volume_IO.Initialize(this->solverOptions);
//	this->volume_IO.readVolume(this->solverOptions->currentIdx);
//
//	this->h_VelocityField = this->volume_IO.getField_float();
//
//	// Copy data to the texture memory
//	this->volumeTexture.setField(h_VelocityField);
//	this->volumeTexture.initialize(Array2Int3(solverOptions->gridSize));
//
//	
//	// Release it from Host
//	volume_IO.release();
//
//
//
//	Vorticity << <blocks, thread >> >
//		(
//			volumeTexture.getTexture(),
//			*this->solverOptions,
//			s_Measure.getSurfaceObject()
//			);
//
//	hipDestroySurfaceObject(this->s_Measure.getSurfaceObject());
//
//}



__host__ bool StreamlineSolver::InitializeVorticityTexture()
{

	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));


	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->a_Measure.getArray();


	// Texture Description
	texDesc.normalizedCoords = true;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.addressMode[0] = hipTextureAddressMode::hipAddressModeBorder;
	texDesc.addressMode[1] = hipTextureAddressMode::hipAddressModeBorder;
	texDesc.addressMode[2] = hipTextureAddressMode::hipAddressModeBorder;
	texDesc.readMode = hipReadModeElementType;


	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_measure, &resDesc, &texDesc, NULL));



	return true;

}