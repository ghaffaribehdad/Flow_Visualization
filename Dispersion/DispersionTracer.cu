#include "hip/hip_runtime.h"
#include "DispersionTracer.h"
#include "DispersionHelper.h"
#include "..//ErrorLogger/ErrorLogger.h"
#include "..//Raycaster/IsosurfaceHelperFunctions.h"
#include <hip/hip_runtime.h>
#include "..//Raycaster/Raycasting_Helper.h"
#include "..//Options/DispresionOptions.h"

//explicit instantiation



bool HeightfieldGenerator::retrace()
{
	//this->heightArray3D.release();
	//this->heightArray3D_extra.release();

	//hipDestroyTextureObject(this->heightFieldTexture3D);
	//hipDestroyTextureObject(this->heightFieldTexture3D_extra);

	//if (!this->InitializeParticles())
	//	return false;

	//// Initialize Height Field as an empty cuda array 3D
	//if (!this->InitializeHeightArray3D())
	//	return false;

	//// Bind the array of heights to the cuda surface
	//if (!this->InitializeHeightSurface3D())
	//	return false;


	//// Trace particle and store their heights on the Height Surface
	//this->trace3D();


	//// Store gradient and height on the surface
	//this->gradient3D();

	////Destroy height + gradient surface and height calculations (both surface and array)
	//this->heightSurface3D.destroySurface();
	//this->heightSurface3D_extra.destroySurface();

	//// Initialize a texture and bind it to height + gradient array
	//if (!this->InitializeHeightTexture3D())
	//	return false;

	return true;
}

bool HeightfieldGenerator::initialize
(
	hipTextureAddressMode addressMode_X ,
	hipTextureAddressMode addressMode_Y ,
	hipTextureAddressMode addressMode_Z 
)
{

	if (!this->initializeRaycastingTexture())				// initilize texture (the texture we need to write to)
		return false;


	if (!this->initializeBoundingBox())		// initialize the bounding box ( copy data to the constant memory of GPU about Bounding Box)
		return false;


	// set the number of rays = number of pixels
	this->rays = (*this->width) * (*this->height);	// Set number of rays based on the number of pixels


	// initialize volume Input Output
	volume_IO.Initialize(this->solverOptions);

	if (!this->InitializeParticles())
		return false;

	// Initialize Height Field as an empty cuda array 3D
	if (!this->InitializeHeightArray3D
	(
		dispersionOptions->gridSize_2D[0],
		dispersionOptions->gridSize_2D[1],
		dispersionOptions->tracingTime
	))
		return false;


	
	// Bind the array of heights to the cuda surface
	if (!this->InitializeHeightSurface3D())
		return false;


	// Trace particle and store their heights on the Height Surface
	this->trace3D_path();
	

	// Store gradient and height on the surface
	this->gradient3D();


	this->heightSurface3D.destroySurface();

	if (!this->InitializeHeightTexture3D())
		return false;

	return true;
}

void HeightfieldGenerator::setResources(Camera* _camera,
	int* _width,
	int* _height,
	SolverOptions* _solverOption,
	RaycastingOptions* _raycastingOptions,
	ID3D11Device* _device,
	IDXGIAdapter* _pAdapter,
	ID3D11DeviceContext* _deviceContext,
	DispersionOptions* _dispersionOptions)
{
	Raycasting::setResources(_camera, _width,_height,_solverOption,_raycastingOptions,_device,_pAdapter,_deviceContext);
		this->dispersionOptions		= _dispersionOptions;
}


__host__ bool HeightfieldGenerator::InitializeParticles()
{
	this->n_particles = dispersionOptions->gridSize_2D[0] * dispersionOptions->gridSize_2D[1];
	this->h_particle = new Particle[dispersionOptions->gridSize_2D[0] * dispersionOptions->gridSize_2D[1]];
	//seedParticle_ZY_Plane(h_particle, solverOptions->gridDiameter, dispersionOptions->gridSize_2D, dispersionOptions->seedWallNormalDist);
	seedParticle_tiltedPlane(h_particle, solverOptions->gridDiameter, dispersionOptions->gridSize_2D, dispersionOptions->seedWallNormalDist, dispersionOptions->tilt_deg);

	size_t Particles_byte = sizeof(Particle) * n_particles;

	gpuErrchk(hipMalloc((void**)& this->d_particle, Particles_byte));
	gpuErrchk(hipMemcpy(this->d_particle, this->h_particle, Particles_byte, hipMemcpyHostToDevice));

	delete[] h_particle;

	return true;
}



__host__ bool HeightfieldGenerator::InitializeHeightArray3D(int x, int y, int z)
{
	// Set dimensions and initialize height field as a 3D CUDA Array
	this->heightArray3D.setDimension(x, y, z);


	this->heightArray3D_extra.setDimension(x, y, z);

	// initialize the 3D array
	if (!heightArray3D.initialize())
		return false;
	if (!heightArray3D_extra.initialize())
		return false;

	return true;
}

__host__ bool HeightfieldGenerator::InitializeHeightArray3D(int3 gridSize)
{
	// Set dimensions and initialize height field as a 3D CUDA Array
	this->heightArray3D.setDimension(gridSize.x, gridSize.y, gridSize.z);


	this->heightArray3D_extra.setDimension(gridSize.x, gridSize.y, gridSize.z);

	// initialize the 3D array
	if (!heightArray3D.initialize())
		return false;
	if (!heightArray3D_extra.initialize())
		return false;

	return true;
}


__host__ bool HeightfieldGenerator::InitializeHeightSurface3D()
{
	// Assign the hightArray to the hightSurface and initialize the surface
	hipArray_t pCudaArray = NULL;
	hipArray_t pCudaArray_extra = NULL;

	pCudaArray = heightArray3D.getArray();
	pCudaArray_extra = heightArray3D_extra.getArray();

	this->heightSurface3D.setInputArray(pCudaArray);
	if (!this->heightSurface3D.initializeSurface())
		return false;

	this->heightSurface3D_extra.setInputArray(pCudaArray_extra);
	if (!this->heightSurface3D_extra.initializeSurface())
		return false;

	return true;
}



// Release resources 
bool HeightfieldGenerator::release()
{
	Raycasting::release();
	hipDestroyTextureObject(this->heightFieldTexture3D);
	this->heightArray3D.release();

	return true;
}

void HeightfieldGenerator::trace3D_path()
{
	// Calculates the block and grid sizes
	unsigned int blocks;
	dim3 thread = { maxBlockDim,maxBlockDim,1 };
	blocks = static_cast<unsigned int>((this->n_particles % (thread.x * thread.y) == 0 ?
		n_particles / (thread.x * thread.y) : n_particles / (thread.x * thread.y) + 1));

	RK4STEP RK4Step = RK4STEP::ODD;
	
	for (int i = 0; i < dispersionOptions->tracingTime; i++)
	{
		if (i == 0)
		{
			// Load i 'dx field in volume_IO into field
			this->LoadVelocityfield(i);
			// Copy and initialize velocityfield texture
			this->initializeVolumeTexuture(hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap, velocityField_0);
			// Release the velocityfield from host (volume_IO)
			volume_IO.release();

			// Same procedure for the second texture
			this->LoadVelocityfield(i+1);
			this->initializeVolumeTexuture(hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap, velocityField_1);
			volume_IO.release();

		}
		else
		{
			// Even integration steps
			if (i % 2 == 0)
			{
				
				this->LoadVelocityfield(i);
				this->velocityField_1.release();
				this->initializeVolumeTexuture(hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap, velocityField_1);
				volume_IO.release();

				RK4Step = RK4STEP::ODD;
			}
			// Odd integration steps
			else
			{
				this->LoadVelocityfield(i);
				this->velocityField_0.release();
				this->initializeVolumeTexuture(hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap, velocityField_0);
				volume_IO.release();

				RK4Step = RK4STEP::EVEN;

			}

		}

		// initialize proper velocityfield

		// trace
		traceDispersion3D_path << < blocks, thread >> >
			(
				d_particle,
				heightSurface3D.getSurfaceObject(),
				heightSurface3D_extra.getSurfaceObject(),
				this->velocityField_0.getTexture(),
				this->velocityField_1.getTexture(),
				*solverOptions,
				*dispersionOptions,
				RK4Step,
				i
			);
	}



	// Calculates the gradients and store it in the cuda surface
	hipFree(d_particle);
}

void HeightfieldGenerator::trace3D()
{
	// Calculates the block and grid sizes
	unsigned int blocks;
	dim3 thread = { maxBlockDim,maxBlockDim,1 };
	blocks = static_cast<unsigned int>((this->n_particles % (thread.x * thread.y) == 0 ?
		n_particles / (thread.x * thread.y) : n_particles / (thread.x * thread.y) + 1));

	// After this step the heightSurface is populated with the height of each particle
	traceDispersion3D_extra << < blocks, thread >> >
		(
			d_particle,
			heightSurface3D.getSurfaceObject(),
			heightSurface3D_extra.getSurfaceObject(),
			this->velocityField_0.getTexture(),
			*solverOptions,
			*dispersionOptions
		);


	// Calculates the gradients and store it in the cuda surface
	hipFree(d_particle);
}


__host__ void HeightfieldGenerator::rendering()
{
	this->deviceContext->PSSetSamplers(0, 1, this->samplerState.GetAddressOf());

	// Create a 2D texture to read hight array

	float bgcolor[] = { 0.0f,0.0f, 0.0f, 1.0f };

	this->deviceContext->ClearRenderTargetView(this->renderTargetView.Get(), bgcolor);// Clear the target view

	// Calculates the block and grid sizes
	unsigned int blocks;
	dim3 thread = { maxBlockDim,maxBlockDim,1 };
	blocks = static_cast<unsigned int>((this->rays % (thread.x * thread.y) == 0 ? rays / (thread.x * thread.y) : rays / (thread.x * thread.y) + 1));
 

	CudaTerrainRenderer_extra<IsosurfaceHelper::Position> << < blocks, thread >> >
		(
			this->raycastingSurface.getSurfaceObject(),
			this->heightFieldTexture3D,
			this->heightFieldTexture3D_extra,
			int(this->rays),
			this->raycastingOptions->samplingRate_0,
			this->raycastingOptions->tolerance_0, 
			*dispersionOptions
		);

}


bool HeightfieldGenerator::updateScene()
{
	if (!this->initializeRaycastingInteroperability())	// Create interoperability while we need to release it at the end of rendering
		return false;

	if (!this->initializeCudaSurface())					// reinitilize cudaSurface	
		return false;

	if (!this->initializeBoundingBox())					//updates constant memory
		return false;


	this->rendering();


	if (!this->raycastingSurface.destroySurface())
		return false;

	this->interoperatibility.release();


	return true;
}





bool HeightfieldGenerator::InitializeHeightTexture3D()
{


	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc viewDes;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&viewDes, 0, sizeof(viewDes));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->heightArray3D.getArray();

	// Texture Description
	texDesc.normalizedCoords = true;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.addressMode[0] = hipTextureAddressMode::hipAddressModeClamp;
	texDesc.addressMode[1] = hipTextureAddressMode::hipAddressModeClamp;
	texDesc.addressMode[2] = hipTextureAddressMode::hipAddressModeClamp;
	texDesc.readMode = hipReadModeElementType;

	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->heightFieldTexture3D, &resDesc, &texDesc, NULL));


	// Use same properties with another array
	//resDesc.res.array.array = this->heightArray3D_extra.getArray();
	//gpuErrchk(hipCreateTextureObject(&this->heightFieldTexture3D_extra, &resDesc, &texDesc, NULL));

	return true;
}







void HeightfieldGenerator::gradient3D()
{

	// Calculates the block and grid sizes
	unsigned int blocks;
	dim3 thread = { maxBlockDim,maxBlockDim,1 };
	blocks = static_cast<unsigned int>((this->n_particles % (thread.x * thread.y) == 0 ?
		n_particles / (thread.x * thread.y) : n_particles / (thread.x * thread.y) + 1));

	// After this step the heightSurface is populated with the height of each particle

	heightFieldGradient3D<IsosurfaceHelper::Position> << < blocks, thread >> >
		(
			heightSurface3D.getSurfaceObject(),
			*dispersionOptions,
			*solverOptions
		);


}

bool HeightfieldGenerator::LoadVelocityfield(const unsigned int& idx)
{

	if (!volume_IO.readVolume(idx))
		return false;

	this->field = volume_IO.flushBuffer_float();

	return true;
}