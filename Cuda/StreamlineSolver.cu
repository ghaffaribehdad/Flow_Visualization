#include "hip/hip_runtime.h"
#include "StreamlineSolver.cuh"
#include "hip/hip_vector_types.h"
// Explicit instantiation
template class StreamlineSolver<float>;
template class StreamlineSolver<double>;

// Kernel of the streamlines, TO-DO: Divide kernel into seprate functions
template <typename T>
__global__ void TracingStream(Particle<T>* d_particles, hipTextureObject_t t_VelocityField, SolverOptions solverOptions, Vertex* p_VertexBuffer)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	if (index < solverOptions.lines_count)
	{
		int lineLength = solverOptions.lineLength;
		int index_buffer = index * lineLength;
		float dt = solverOptions.dt;
		float3 gridDiameter =
		{
			solverOptions.gridDiameter[0],
			solverOptions.gridDiameter[1],
			solverOptions.gridDiameter[2]
		};

		int3 gridSize =
		{
			solverOptions.gridSize[0],
			solverOptions.gridSize[1],
			solverOptions.gridSize[2]
		};

		for (int i = 0; i < lineLength; i++)
		{
			d_particles[index].move(dt, gridSize, gridDiameter, t_VelocityField);

			p_VertexBuffer[index_buffer + i].pos.x = d_particles[index].getPosition()->x - gridDiameter.x / 2.0;
			p_VertexBuffer[index_buffer + i].pos.y = d_particles[index].getPosition()->y - gridDiameter.y / 2.0;
			p_VertexBuffer[index_buffer + i].pos.z = d_particles[index].getPosition()->z + gridDiameter.z / 2.0;
			float3* velocity = d_particles[index].getVelocity();
			float3 norm = normalize(*velocity);
			p_VertexBuffer[index_buffer + i].tangent.x = norm.x;
			p_VertexBuffer[index_buffer + i].tangent.y = norm.y;
			p_VertexBuffer[index_buffer + i].tangent.z = norm.z;
			p_VertexBuffer[index_buffer + i].LineID = float(index) / float(solverOptions.lines_count);


			switch (solverOptions.colorMode)
			{
				case 0: // Velocity
				{
				
					p_VertexBuffer[index_buffer + i].LineID = float(index) / float(solverOptions.lines_count);

				}
				case 1: // Vx
				{
					float velocity = d_particles[index].getVelocity()->x;
					p_VertexBuffer[index_buffer + i].color.x = velocity;

				}
				case 2: // Vx
				{
					float velocity = d_particles[index].getVelocity()->y;
					p_VertexBuffer[index_buffer + i].color.x = velocity;
				}
				case 3: // Vx
				{
					float velocity = d_particles[index].getVelocity()->z;
					p_VertexBuffer[index_buffer + i].color.x = velocity;
				}
			}

			
		}
	}

}
template <typename T>
__host__ void StreamlineSolver<T>::release()
{
	hipFree(this->d_Particles);
	hipFree(this->d_VelocityField);
	hipDestroyTextureObject(this->t_VelocityField);
}

template <typename T>
__host__ bool StreamlineSolver<T>::solve()
{
	this->volume_IO.Initialize(this->solverOptions);

	// TO-DO: Define streamlinesolver for double precision
	this->h_VelocityField = InitializeVelocityField(this->solverOptions.currentIdx);
	this->InitializeTexture(h_VelocityField, t_VelocityField);

	volume_IO.release();
	this->InitializeParticles();
	
	int blockDim = 256;
	int thread = (this->solverOptions.lines_count / blockDim)+1;
	
	TracingStream<T> << <blockDim , thread >> > (this->d_Particles, t_VelocityField, solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer));

	this->release();

	return true;
}