#include "cudaSurface.h"
#include "../ErrorLogger/ErrorLogger.h"

bool CudaSurface::initializeSurface()
{
	// Specify surface
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;

	// Create the surface objects
	resDesc.res.array.array = this->cuInputArray;

	gpuErrchk(hipCreateSurfaceObject(&surfaceObject, &resDesc));

	return true;
}


bool CudaSurface::destroySurface()
{

	// Destroy surface objects
	gpuErrchk(hipDestroySurfaceObject(surfaceObject));

	

	return true;
}
