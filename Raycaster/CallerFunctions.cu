#include "hip/hip_runtime.h"
#pragma once
#include "CallerFunctions.h"
#include "IsosurfaceHelperFunctions.h"


__device__ float callerValueAtTex(int i, hipTextureObject_t tex, float3 position, float3  gridDiameter, int3 gridSize) {
	switch (i)
	{
	case IsoMeasure::Velocity_X:
		return Channel_X::ValueAtXYZ(tex, position);
		break;

	case IsoMeasure::Velocity_Y:
		return Channel_Y::ValueAtXYZ(tex, position);
		break;

	case IsoMeasure::Velocity_Z:
		return Channel_Z::ValueAtXYZ(tex, position);
		break;

	case IsoMeasure::Velocity_W:
		return Channel_W::ValueAtXYZ(tex, position);
		break;

	case IsoMeasure::VelocityMagnitude:
		return Velocity_Magnitude::ValueAtXYZ(tex, position);
		break;

	case IsoMeasure::ShearStress:
		return ShearStress::ValueAtXYZ(tex, position);
		break;
		
	case IsoMeasure::LAMBDA2:
		return Lambda2::ValueAtXYZ(tex, position, gridDiameter, gridSize);
		break;
	}

	return 0;
}


__device__ float3 callerGradientAtTex(int i, hipTextureObject_t tex, const float3 & position, const float3 & gridDiameter, const int3 & gridSize)
{
	switch (i)
	{
	case IsoMeasure::Velocity_X:
		return Channel_X::GradientAtXYZ_Tex(tex, position, gridDiameter, gridSize);
		break;

	case IsoMeasure::Velocity_Y:
		return Channel_Y::GradientAtXYZ_Tex(tex, position, gridDiameter, gridSize);
		break;

	case IsoMeasure::Velocity_Z:
		return Channel_Z::GradientAtXYZ_Tex(tex, position, gridDiameter, gridSize);
		break;

	case IsoMeasure::Velocity_W:
		return Channel_W::GradientAtXYZ_Tex(tex, position, gridDiameter, gridSize);
		break;

	case IsoMeasure::VelocityMagnitude:
		return Velocity_Magnitude::GradientAtXYZ_Tex(tex, position, gridDiameter, gridSize);
		break;

	case IsoMeasure::ShearStress:
		return ShearStress::GradientAtXYZ_Tex(tex, position, gridDiameter, gridSize);
		break;

	case IsoMeasure::LAMBDA2:
		return Lambda2::GradientAtXYZ_Tex(tex, position, gridDiameter, gridSize);
		break;
	}
	return { 0,0,0 };
}
