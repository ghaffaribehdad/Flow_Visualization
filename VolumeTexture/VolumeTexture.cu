#include "hip/hip_runtime.h"

#include "VolumeTexture.h"
#include <string>






hipTextureObject_t VolumeTexture::initialize()
{
	// Cuda 3D array of velocities
	hipArray_t cuArray_velocity;


	// define the size of the velocity field
	hipExtent extent =
	{
		static_cast<size_t>(this->solverOptions->gridSize[0]),
		static_cast<size_t>(this->solverOptions->gridSize[1]),
		static_cast<size_t>(this->solverOptions->gridSize[2])
	};


	// Allocate 3D Array
	hipChannelFormatDesc channelFormatDesc = hipCreateChannelDesc<float4>();
	hipMalloc3DArray(&cuArray_velocity, &channelFormatDesc, extent, 0);



	// set copy parameters to copy from velocity field to array
	hipMemcpy3DParms cpyParams = { 0 };

	cpyParams.srcPtr = make_hipPitchedPtr((void*)this->h_field, extent.width * sizeof(float4), extent.height, extent.depth);
	cpyParams.dstArray = cuArray_velocity;
	cpyParams.kind = hipMemcpyHostToDevice;
	cpyParams.extent = extent;


	// Copy velocities to 3D Array
	gpuErrchk(hipMemcpy3D(&cpyParams));
	// might need sync before release the host memory

	// Release the Volume while it is copied on GPU
	//this->volume_IO.release();


	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray_velocity;

	// Texture Description
	texDesc.normalizedCoords = true;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.addressMode[0] = hipAddressModeBorder;
	texDesc.addressMode[1] = hipAddressModeBorder;
	texDesc.addressMode[2] = hipAddressModeBorder;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return t_field;

}
void VolumeTexture::release()
{
	hipDestroyTextureObject(this->t_field);
}



