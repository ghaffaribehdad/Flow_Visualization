#include "hip/hip_runtime.h"
#include "Raycasting.h"
#include "IsosurfaceHelperFunctions.h"
#include "Raycasting_Helper.h"
#include "hip/hip_runtime.h"
#include "..//Cuda/CudaHelperFunctions.h"
#include "..//Options/DispresionOptions.h"

__constant__ BoundingBox d_boundingBox;
__constant__ float3 d_raycastingColor;

// Explicit instantiation
template __global__ void CudaIsoSurfacRenderer<struct IsosurfaceHelper::Velocity_Magnitude>	(hipSurfaceObject_t raycastingSurface, hipTextureObject_t field1, int rays, float isoValue, float samplingRate, float IsosurfaceTolerance);
template __global__ void CudaIsoSurfacRenderer<struct IsosurfaceHelper::Velocity_X>			(hipSurfaceObject_t raycastingSurface, hipTextureObject_t field1, int rays, float isoValue, float samplingRate, float IsosurfaceTolerance);
template __global__ void CudaIsoSurfacRenderer<struct IsosurfaceHelper::Velocity_Y>			(hipSurfaceObject_t raycastingSurface, hipTextureObject_t field1, int rays, float isoValue, float samplingRate, float IsosurfaceTolerance);
template __global__ void CudaIsoSurfacRenderer<struct IsosurfaceHelper::Velocity_Z>			(hipSurfaceObject_t raycastingSurface, hipTextureObject_t field1, int rays, float isoValue, float samplingRate, float IsosurfaceTolerance);
template __global__ void CudaIsoSurfacRenderer<struct IsosurfaceHelper::ShearStress>		(hipSurfaceObject_t raycastingSurface, hipTextureObject_t field1, int rays, float isoValue, float samplingRate, float IsosurfaceTolerance);
template __global__ void CudaTerrainRenderer< struct IsosurfaceHelper::Position >			(hipSurfaceObject_t raycastingSurface, hipTextureObject_t field1, int rays, float samplingRate, float IsosurfaceTolerance, DispersionOptions dispersionOptions);


__host__ bool Raycasting::updateScene()
{
	if (!this->initializeRaycastingInteroperability())	// Create interoperability while we need to release it at the end of rendering
		return false;

	if (!this->initializeCudaSurface())					// reinitilize cudaSurface	
		return false;

	if (!this->initializeBoundingBox())					//updates constant memory
		return false;

	this->rendering();


	if (!this->raycastingSurface.destroySurface())
		return false;

	this->interoperatibility.release();

	return true;

}

__host__ bool Raycasting::resize()
{
	this->raycastingTexture->Release();
	this->initializeRaycastingTexture();

	this->raycastingSurface.destroySurface();
	this->interoperatibility.release();

	this->initializeRaycastingInteroperability();
	this->initializeCudaSurface();
	this->initializeBoundingBox();
	this->rendering();
	this->interoperatibility.release();

	return true;
}

__host__ bool Raycasting::initialize
(
	hipTextureAddressMode addressMode_X = hipAddressModeBorder,
	hipTextureAddressMode addressMode_Y = hipAddressModeBorder,
	hipTextureAddressMode addressMode_Z = hipAddressModeBorder
)
{
	if (!this->initializeRaycastingTexture())				// initilize texture (the texture we need to write to)
		return false;


	if (!this->initializeBoundingBox())		// initialize the bounding box ( copy data to the constant memory of GPU about Bounding Box)
		return false;

	// set the number of rays = number of pixels
	this->rays = (*this->width) * (*this->height);	// Set number of rays based on the number of pixels


	// Read and set field
	if(!this->raycastingOptions->fileLoaded)		// Load data set into the texture memory
	{
		this->volume_IO.Initialize(this->solverOptions);
		this->initializeIO();
		this->initializeVolumeTexuture(addressMode_X, addressMode_Y, addressMode_Z);

		this->raycastingOptions->fileLoaded = true;
	}
	if (this->raycastingOptions->fileChanged)
	{
		this->initializeIO();
		this->volumeTexture.release();
		this->initializeVolumeTexuture(addressMode_X, addressMode_Y, addressMode_Z);

		this->raycastingOptions->fileChanged = false;
	}

	return true;

}

__host__ bool Raycasting::release()
{
	this->interoperatibility.release();
	this->volumeTexture.release();
	this->raycastingSurface.destroySurface();

	return true;
}

__host__ void Raycasting::rendering()
{

	this->deviceContext->PSSetSamplers(0, 1, this->samplerState.GetAddressOf());



	float bgcolor[] = { 0.0f,0.0f, 0.0f, 1.0f };

	this->deviceContext->ClearRenderTargetView(this->renderTargetView.Get(), bgcolor);// Clear the target view

	// Calculates the block and grid sizes
	unsigned int blocks;
	dim3 thread = { 32,32,1 };
	blocks = static_cast<unsigned int>((this->rays % (thread.x * thread.y) == 0 ? rays / (thread.x * thread.y) : rays / (thread.x * thread.y) + 1));




	// Optimize blocks and grid sizes
	//int* minGridSize	= nullptr;
	//int* blockSize		= nullptr;	
	//hipModuleOccupancyMaxPotentialBlockSize(minGridSize, blockSize,(hipFunction_t)CudaIsoSurfacRenderer<IsosurfaceHelper::Velocity_Magnitude>, 0, 0,0);



	// TODO:
	// Alternatively use ENUM templates! 
	switch (this->raycastingOptions->isoMeasure_0)
	{
		case IsoMeasure::VelocityMagnitude:
		{
			CudaIsoSurfacRenderer<IsosurfaceHelper::Velocity_Magnitude> << < blocks, thread >> >
				(
					this->raycastingSurface.getSurfaceObject(),
					this->volumeTexture.getTexture(),
					int(this->rays),
					this->raycastingOptions->isoValue_0,
					this->raycastingOptions->samplingRate_0,
					this->raycastingOptions->tolerance_0
					);
			break;
		}

		case IsoMeasure::Velocity_x:
		{
			CudaIsoSurfacRenderer<IsosurfaceHelper::Velocity_X> << < blocks, thread >> >
				(
					this->raycastingSurface.getSurfaceObject(),
					this->volumeTexture.getTexture(),
					int(this->rays),
					this->raycastingOptions->isoValue_0,
					this->raycastingOptions->samplingRate_0,
					this->raycastingOptions->tolerance_0
				);
			break;

		}

		case IsoMeasure::Velocity_y:
		{
			CudaIsoSurfacRenderer<IsosurfaceHelper::Velocity_Y> << < blocks, thread >> >
				(
					this->raycastingSurface.getSurfaceObject(),
					this->volumeTexture.getTexture(),
					int(this->rays),
					this->raycastingOptions->isoValue_0,
					this->raycastingOptions->samplingRate_0,
					this->raycastingOptions->tolerance_0
				);
			break;
		}

		case IsoMeasure::Velocity_Z:
		{
			CudaIsoSurfacRenderer<IsosurfaceHelper::Velocity_Z> << < blocks, thread >> >
				(
					this->raycastingSurface.getSurfaceObject(),
					this->volumeTexture.getTexture(),
					int(this->rays),
					this->raycastingOptions->isoValue_0,
					this->raycastingOptions->samplingRate_0,
					this->raycastingOptions->tolerance_0
				);
			break;
		}

		case IsoMeasure::ShearStress:
		{
			CudaIsoSurfacRenderer<IsosurfaceHelper::ShearStress> << < blocks, thread >> >
				(
					this->raycastingSurface.getSurfaceObject(),
					this->volumeTexture.getTexture(),
					int(this->rays),
					this->raycastingOptions->isoValue_0,
					this->raycastingOptions->samplingRate_0,
					this->raycastingOptions->tolerance_0
				);
			break;
		}

	}



}



__host__ bool Raycasting::initializeBoundingBox()
{

	BoundingBox * h_boundingBox = new BoundingBox;

	h_boundingBox->eyePos = XMFloat3ToFloat3(camera->GetPositionFloat3());
	h_boundingBox->viewDir = XMFloat3ToFloat3(camera->GetViewVector());
	h_boundingBox->upVec = XMFloat3ToFloat3(camera->GetUpVector());


	// Multiply and store Projectiopn and View Matrix in View Matrix
	
	h_boundingBox->width = *width;
	h_boundingBox->height= *height;
	h_boundingBox->gridDiameter = ArrayFloat3ToFloat3(solverOptions->gridDiameter);
	h_boundingBox->gridSize = ArrayInt3ToInt3(solverOptions->gridSize);
	h_boundingBox->updateBoxFaces();
	h_boundingBox->updateAspectRatio();
	h_boundingBox->constructEyeCoordinates();
	h_boundingBox->FOV = (this->FOV_deg / 360.0f)* XM_2PI;
	h_boundingBox->distImagePlane = this->distImagePlane;

	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_boundingBox), h_boundingBox, sizeof(BoundingBox)));
	
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_raycastingColor), this->raycastingOptions->color_0, sizeof(float3)));


	delete h_boundingBox;
	
	return true;
}


__host__ bool Raycasting::initializeVolumeTexuture
(
	hipTextureAddressMode addressMode_X ,
	hipTextureAddressMode addressMode_Y ,
	hipTextureAddressMode addressMode_Z
	)
{
	this->volumeTexture.setSolverOptions(this->solverOptions);
	this->volumeTexture.setField(this->field);
	this->volumeTexture.initialize
	(
		addressMode_X,
		addressMode_Y,
		addressMode_Z
	);

	return true;
}

__host__ bool Raycasting::initializeIO()
{
	
	this->volume_IO.readVolume(this->solverOptions->currentIdx);
	std::vector<char>* p_vec_buffer = volume_IO.flushBuffer();
	char* p_vec_buffer_temp = &(p_vec_buffer->at(0));
	this->field = reinterpret_cast<float*>(p_vec_buffer_temp);
	
	return true;
}


 




template <typename Observable>
__global__ void CudaIsoSurfacRenderer
(
	hipSurfaceObject_t raycastingSurface,
	hipTextureObject_t field1,
	int rays, float isoValue,
	float samplingRate,
	float IsosurfaceTolerance
)
{

	Observable observable;

	int index = blockIdx.x * blockDim.y * blockDim.x;
	index += threadIdx.y * blockDim.x;
	index += threadIdx.x;

	if (index < rays)
	{

		// determine pixel position based on the index of the thread
		int2 pixel;
		pixel.y = index / d_boundingBox.width;
		pixel.x = index - pixel.y * d_boundingBox.width;

		// copy values from constant memory to local memory (which one is faster?)
		float3 viewDir = d_boundingBox.viewDir;
		float3 pixelPos = pixelPosition(d_boundingBox, pixel.x, pixel.y);
		float2 NearFar = findIntersections(pixelPos, d_boundingBox);

		
		// if inside the bounding box
		if (NearFar.y != -1)
		{

			float3 rayDir = normalize(pixelPos - d_boundingBox.eyePos);

			// near and far plane
			float n = 0.1f;
			float f = 1000.0f;

			// Add the offset to the eye position
			float3 eyePos = d_boundingBox.eyePos + d_boundingBox.gridDiameter / 2.0;

			for (float t = NearFar.x; t < NearFar.y; t = t + samplingRate)
			{
				// Position of the isosurface
				float3 position = pixelPos + (rayDir * t);

				// Adds an offset to position while the center of the grid is at gridDiamter/2
				position += d_boundingBox.gridDiameter / 2.0;

				

				//Relative position calculates the position of the point on the cuda texture
				float3 relativePos = (position / d_boundingBox.gridDiameter);




				// check if we have a hit 
				if (observable.ValueAtXYZ(field1, relativePos) - isoValue > 0)
				{

					position = binarySearch<Observable>(observable, field1, position, d_boundingBox.gridDiameter, rayDir * t, isoValue, IsosurfaceTolerance, 50);
					relativePos = (position / d_boundingBox.gridDiameter);

					// calculates gradient
					float3 gradient = observable.GradientAtGrid(field1, relativePos, d_boundingBox.gridSize);

					// shading (no ambient)
					float diffuse = max(dot(normalize(gradient), viewDir), 0.0f);
					float3 rgb = d_raycastingColor * diffuse;


					// vector from eye to isosurface
					float3 position_viewCoordinate = position - eyePos;

					// calculates the z-value
					float z_dist = abs(dot(viewDir, position_viewCoordinate));

					// calculate non-linear depth between 0 to 1
					float depth = (f) / (f - n);
					depth += (-1.0f / z_dist) * (f * n) / (f - n);

					float4 rgba = { rgb.x, rgb.y, rgb.z, depth};
					
					// write back color and depth into the texture (surface)
					// stride size of 4 * floats for each texel
					surf2Dwrite(rgba, raycastingSurface, 4 * sizeof(float) * pixel.x, pixel.y);
					break;
				}
				

			}



		}

	}


}


template <typename Observable>
__global__ void CudaTerrainRenderer
(
	hipSurfaceObject_t raycastingSurface,
	hipTextureObject_t field1,
	int rays,
	float samplingRate,
	float IsosurfaceTolerance,
	DispersionOptions dispersionOptions
)
{
	Observable observable;

	int index = blockIdx.x * blockDim.y * blockDim.x;
	index += threadIdx.y * blockDim.x;
	index += threadIdx.x;

	if (index < rays)
	{

		// determine pixel position based on the index of the thread
		int2 pixel;
		pixel.y = index / d_boundingBox.width;
		pixel.x = index - pixel.y * d_boundingBox.width;

		// copy values from constant memory to local memory (which one is faster?)
		float3 viewDir = d_boundingBox.viewDir;
		float3 pixelPos = pixelPosition(d_boundingBox, pixel.x, pixel.y);
		float2 NearFar = findIntersections(pixelPos, d_boundingBox);


		// if inside the bounding box
		if (NearFar.y != -1)
		{

			float3 rayDir = normalize(pixelPos - d_boundingBox.eyePos);

			// near and far plane
			float n = 0.1f;
			float f = 1000.0f;

			// Add the offset to the eye position
			float3 eyePos = d_boundingBox.eyePos + d_boundingBox.gridDiameter / 2.0;

			for (float t = NearFar.x; t < NearFar.y; t = t + samplingRate)
			{
				// Position of the isosurface
				float3 position = pixelPos + (rayDir * t);

				// Adds an offset to position while the center of the grid is at gridDiamter/2
				position += d_boundingBox.gridDiameter / 2.0;



				//Relative position calculates the position of the point on the cuda texture
				float2 relativePos = make_float2(position.x / d_boundingBox.gridDiameter.x, position.z / d_boundingBox.gridDiameter.z);
				
				// fetch texels from the GPU memory
				float4 hightFieldVal = observable.ValueAtXY(field1, relativePos);

				// check if we have a hit 
				if (position.y - hightFieldVal.x > 0 &&  position.y - hightFieldVal.x < 0.01 )
				{

					float3 samplingStep = rayDir * samplingRate;
					//binary search
					position = binarySearch_heightField
					(
						position,
						field1,
						samplingStep,
						d_boundingBox.gridDiameter,
						dispersionOptions.binarySearchTolerance,
						dispersionOptions.binarySearchMaxIteration
					);

					relativePos = make_float2(position.x / d_boundingBox.gridDiameter.x, position.z / d_boundingBox.gridDiameter.z);

					hightFieldVal = observable.ValueAtXY(field1, relativePos);

					float3 gradient = { -hightFieldVal.y,-1,-hightFieldVal.z };


					// shading (no ambient)
					float diffuse = max(dot(normalize(gradient), viewDir), 0.0f);

					float3 rgb = { 0,1,0 };


					rgb = rgb * diffuse;

					// vector from eye to isosurface
					float3 position_viewCoordinate = position - eyePos;

					// calculates the z-value
					float z_dist = abs(dot(viewDir, position_viewCoordinate));

					// calculate non-linear depth between 0 to 1
					float depth = (f) / (f - n);
					depth += (-1.0f / z_dist) * (f * n) / (f - n);

					float4 rgba = { rgb.x , rgb.y, rgb.z, depth };

					// write back color and depth into the texture (surface)
					// stride size of 4 * floats for each texel
					surf2Dwrite(rgba, raycastingSurface, 4 * sizeof(float) * pixel.x, pixel.y);
					break;
				}


			}



		}

	}


}

bool Raycasting::initializeRaycastingTexture()
{
	D3D11_TEXTURE2D_DESC textureDesc;
	ZeroMemory(&textureDesc, sizeof(textureDesc));

	textureDesc.ArraySize = 1;
	textureDesc.BindFlags = D3D11_BIND_RENDER_TARGET | D3D11_BIND_SHADER_RESOURCE;
	textureDesc.CPUAccessFlags = 0;
	textureDesc.Format = DXGI_FORMAT_R32G32B32A32_FLOAT;
	textureDesc.Height = *this->height;
	textureDesc.Width = *this->width;
	textureDesc.MipLevels = 2;
	textureDesc.MiscFlags = 0;
	textureDesc.SampleDesc.Count = 1;
	textureDesc.SampleDesc.Quality = 0;
	textureDesc.Usage = D3D11_USAGE_DEFAULT;


	HRESULT hr = this->device->CreateTexture2D(&textureDesc, nullptr, this->raycastingTexture.GetAddressOf());
	if (FAILED(hr))
	{
		ErrorLogger::Log(hr, "Failed to Create Front Texture");
	}

	// Create Render targe view
	hr = this->device->CreateRenderTargetView(raycastingTexture.Get(), NULL, this->renderTargetView.GetAddressOf());
	if (FAILED(hr))
	{
		ErrorLogger::Log(hr, "Failed to Create RenderTargetView");
		return false;
	}

	return true;

}

bool Raycasting::initializeRaycastingInteroperability()
{
	// define interoperation descriptor and set it to zero
	Interoperability_desc interoperability_desc;
	memset(&interoperability_desc, 0, sizeof(interoperability_desc));

	// set interoperation descriptor
	interoperability_desc.flag = hipGraphicsRegisterFlagsSurfaceLoadStore;
	interoperability_desc.p_adapter = this->pAdapter;
	interoperability_desc.p_device = this->device;
	//interoperability_desc.size = sizeof(float) * static_cast<size_t>(*this->width) * static_cast<size_t>(*this->height);
	interoperability_desc.size = (size_t)4.0 * sizeof(float) * static_cast<size_t>(*this->width) * static_cast<size_t>(*this->height);
	interoperability_desc.pD3DResource = this->raycastingTexture.Get();

	// initialize the interoperation
	this->interoperatibility.setInteroperability_desc(interoperability_desc);

	return this->interoperatibility.Initialize();
}

__host__ bool Raycasting::initializeCudaSurface()
{

	hipArray_t pCudaArray = NULL;


	// Get the cuda Array from the interoperability ( array to the texture)
	this->interoperatibility.getMappedArray(pCudaArray);

	// Pass this cuda Array to the raycasting Surface
	this->raycastingSurface.setInputArray(pCudaArray);

	//this->raycastingSurface.setDimensions(*this->width, *this->height);

	// Create cuda surface 
	if (!this->raycastingSurface.initializeSurface())
		return false;

	// To release we need to destory surface and free the cuda array kept in the interpoly

	return true;
}


__host__ void Raycasting::setResources
(
	Camera* _camera,
	int* _width,
	int* _height,
	SolverOptions* _solverOption,
	RaycastingOptions* _raycastingOptions,
	ID3D11Device* _device,
	IDXGIAdapter* _pAdapter,
	ID3D11DeviceContext* _deviceContext
)
{
	this->camera = _camera;
	this->FOV_deg = 30.0;
	this->width = _width;
	this->height = _height;

	this->solverOptions = _solverOption;
	this->raycastingOptions = _raycastingOptions;

	this->device = _device;
	this->pAdapter = _pAdapter;
	this->deviceContext = _deviceContext;
}


bool Raycasting::initializeShaders()
{
	if (this->vertexBuffer.Get() == nullptr)
	{
		std::wstring shaderfolder;
#pragma region DetermineShaderPath
		if (IsDebuggerPresent() == TRUE)
		{
#ifdef _DEBUG //Debug Mode
#ifdef _WIN64 //x64
			shaderfolder = L"x64\\Debug\\";
#else //x86
			shaderfolder = L"Debug\\"
#endif // DEBUG
#else //Release mode
#ifdef _WIN64 //x64
			shaderfolder = L"x64\\Release\\";
#else  //x86
			shaderfolder = L"Release\\"
#endif // Release
#endif // _DEBUG or Release mode
		}

		D3D11_INPUT_ELEMENT_DESC layout[] =
		{
			{
				"POSITION",
				0,
				DXGI_FORMAT::DXGI_FORMAT_R32G32B32_FLOAT,
				0,
				D3D11_APPEND_ALIGNED_ELEMENT,
				D3D11_INPUT_CLASSIFICATION::D3D11_INPUT_PER_VERTEX_DATA,
				0
			},

			{
				"TEXCOORD",
				0,
				DXGI_FORMAT::DXGI_FORMAT_R32G32_FLOAT,
				0, 
				D3D11_APPEND_ALIGNED_ELEMENT,
				D3D11_INPUT_CLASSIFICATION::D3D11_INPUT_PER_VERTEX_DATA,
				0 
			}
		};

		UINT numElements = ARRAYSIZE(layout);

		if (!vertexshader.Initialize(this->device, shaderfolder + L"vertexshaderTexture.cso", layout, numElements))
			return false;

		if (!pixelshader.Initialize(this->device, shaderfolder + L"pixelshaderTextureSampler.cso"))
			return false;
	}


	return true;
}


bool Raycasting::initializeScene()
{
	if (vertexBuffer.Get() == nullptr)
	{
		TexCoordVertex BoundingBox[] =
		{
			TexCoordVertex(-1.0f,	-1.0f,	1.0f,	0.0f,	1.0f), //Bottom Left 
			TexCoordVertex(-1.0f,	1.0f,	1.0f,	0.0f,	0.0f), //Top Left
			TexCoordVertex(1.0f,	1.0f,	1.0f,	1.0f,	0.0f), //Top Right

			TexCoordVertex(-1.0f,	-1.0f,	1.0f,	0.0f,	1.0f), //Bottom Left 
			TexCoordVertex(1.0f,	1.0f,	1.0f,	1.0f,	0.0f), //Top Right
			TexCoordVertex(1.0f,	-1.0f,	1.0f,	1.0f,	1.0f), //Bottom Right

		};


		this->vertexBuffer.Initialize(this->device, BoundingBox, ARRAYSIZE(BoundingBox));
	}



	return true;
}




bool Raycasting::createRaycastingShaderResourceView()
{

	if (shaderResourceView == nullptr)
	{
		D3D11_SHADER_RESOURCE_VIEW_DESC shader_resource_view_desc;
		ZeroMemory(&shader_resource_view_desc, sizeof(shader_resource_view_desc));

		shader_resource_view_desc.Format = DXGI_FORMAT_R32G32B32A32_FLOAT;
		shader_resource_view_desc.Texture2D.MipLevels = 2;
		shader_resource_view_desc.ViewDimension = D3D11_SRV_DIMENSION_TEXTURE2D;

		HRESULT hr = this->device->CreateShaderResourceView(
			this->getTexture(),
			&shader_resource_view_desc,
			shaderResourceView.GetAddressOf()
		);

		if (FAILED(hr))
		{
			ErrorLogger::Log(hr, "Failed to Create shader resource view");
			return false;
		}
	}


	return true;
}


bool Raycasting::initializeSamplerstate()
{
	if (samplerState.Get() == nullptr)
	{
		//Create sampler description for sampler state
		D3D11_SAMPLER_DESC sampDesc;
		ZeroMemory(&sampDesc, sizeof(sampDesc));
		sampDesc.Filter = D3D11_FILTER_MIN_MAG_MIP_LINEAR;
		sampDesc.AddressU = D3D11_TEXTURE_ADDRESS_CLAMP;
		sampDesc.AddressV = D3D11_TEXTURE_ADDRESS_CLAMP;
		sampDesc.AddressW = D3D11_TEXTURE_ADDRESS_CLAMP;
		sampDesc.ComparisonFunc = D3D11_COMPARISON_NEVER;
		sampDesc.MinLOD = 0;
		sampDesc.MaxLOD = D3D11_FLOAT32_MAX;
		HRESULT hr = this->device->CreateSamplerState(&sampDesc, this->samplerState.GetAddressOf()); //Create sampler state
		if (FAILED(hr))
		{
			ErrorLogger::Log(hr, "Failed to create sampler state.");
			return false;
		}
	}


	return true;
}



bool Raycasting::initializeRasterizer()
{
	if (this->rasterizerstate.Get() == nullptr)
	{
		// Create Rasterizer state
		D3D11_RASTERIZER_DESC rasterizerDesc;
		ZeroMemory(&rasterizerDesc, sizeof(D3D11_RASTERIZER_DESC));

		rasterizerDesc.FillMode = D3D11_FILL_MODE::D3D11_FILL_SOLID;
		rasterizerDesc.CullMode = D3D11_CULL_MODE::D3D11_CULL_NONE; // CULLING could be set to none
		rasterizerDesc.MultisampleEnable = false;
		rasterizerDesc.AntialiasedLineEnable = false;
		//rasterizerDesc.FrontCounterClockwise = TRUE;//= 1;

		HRESULT hr = this->device->CreateRasterizerState(&rasterizerDesc, this->rasterizerstate.GetAddressOf());
		if (FAILED(hr))
		{
			ErrorLogger::Log(hr, "Failed to Create rasterizer state.");
			return false;
		}

	}

	return true;
}

void Raycasting::setShaders()
{

	this->deviceContext->IASetInputLayout(this->vertexshader.GetInputLayout());		// Set the input layout

	// set the primitive topology
	this->deviceContext->IASetPrimitiveTopology(D3D11_PRIMITIVE_TOPOLOGY::D3D_PRIMITIVE_TOPOLOGY_TRIANGLELIST);		

	this->deviceContext->RSSetState(this->rasterizerstate.Get());					// set the rasterizer state
	this->deviceContext->VSSetShader(vertexshader.GetShader(), NULL, 0);			// set vertex shader
	this->deviceContext->PSSetShader(pixelshader.GetShader(), NULL, 0);
	UINT offset = 0;

	// set Vertex buffer
	this->deviceContext->IASetVertexBuffers(0, 1, this->vertexBuffer.GetAddressOf(), this->vertexBuffer.StridePtr(), &offset); 
	this->deviceContext->PSSetSamplers(0, 1, this->samplerState.GetAddressOf());
	this->deviceContext->PSSetShaderResources(0, 1, this->shaderResourceView.GetAddressOf());
}


void Raycasting::draw()
{
	this->initializeRasterizer();
	this->initializeSamplerstate();
	this->createRaycastingShaderResourceView();

	this->initializeShaders();
	this->initializeScene();


	this->setShaders();
	this->deviceContext->Draw(6, 0);
}