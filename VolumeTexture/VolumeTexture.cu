#include "hip/hip_runtime.h"

#include "VolumeTexture.h"
#include <string>






bool VolumeTexture3D::initialize
(
	hipTextureAddressMode addressMode_x ,
	hipTextureAddressMode addressMode_y ,
	hipTextureAddressMode addressMode_z,
	hipTextureFilterMode _cudaTextureFilterMode
)
{
	if (this->solverOptions == nullptr)
	{
		return false;
	}

	hipExtent extent = make_hipExtent(this->solverOptions->gridSize[0], this->solverOptions->gridSize[1], this->solverOptions->gridSize[2]);

	// Allocate 3D Array
	hipChannelFormatDesc channelFormatDesc = hipCreateChannelDesc<float4>();
	hipMalloc3DArray(&this->cuArray_velocity , &channelFormatDesc, extent);



	// set copy parameters to copy from velocity field to array
	hipMemcpy3DParms cpyParams = { 0 };

	cpyParams.srcPtr = make_hipPitchedPtr((void*)this->h_field,extent.width * sizeof(float4),extent.width, extent.height);
	cpyParams.dstArray = this->cuArray_velocity;
	cpyParams.kind = hipMemcpyHostToDevice;
	cpyParams.extent = extent;
	
	// Copy velocities to 3D Array
	gpuErrchk(hipMemcpy3D(&cpyParams));
	// might need sync before release the host memory


	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description
	
	texDesc.filterMode = _cudaTextureFilterMode;
	if (_cudaTextureFilterMode == hipFilterModeLinear)
	{
		texDesc.normalizedCoords = true;
	}
	else
	{
		texDesc.normalizedCoords = false;
	}

	texDesc.addressMode[0] = addressMode_x;
	texDesc.addressMode[1] = addressMode_y;
	texDesc.addressMode[2] = addressMode_z;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return true;

}


bool VolumeTexture3D::initialize
(
	int3 dimension,
	hipTextureAddressMode addressMode_x,
	hipTextureAddressMode addressMode_y,
	hipTextureAddressMode addressMode_z,
	hipTextureFilterMode _cudaTextureFilterMode
)
{

	hipExtent extent = make_hipExtent(dimension.x, dimension.y, dimension.z);

	// Allocate 3D Array
	hipChannelFormatDesc channelFormatDesc = hipCreateChannelDesc<float4>();
	hipMalloc3DArray(&this->cuArray_velocity, &channelFormatDesc, extent);



	// set copy parameters to copy from velocity field to array
	hipMemcpy3DParms cpyParams = { 0 };

	cpyParams.srcPtr = make_hipPitchedPtr((void*)this->h_field, extent.width * sizeof(float4), extent.width, extent.height);
	cpyParams.dstArray = this->cuArray_velocity;
	cpyParams.kind = hipMemcpyHostToDevice;
	cpyParams.extent = extent;

	// Copy velocities to 3D Array
	gpuErrchk(hipMemcpy3D(&cpyParams));
	// might need sync before release the host memory


	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description
	texDesc.filterMode = _cudaTextureFilterMode;
	if (_cudaTextureFilterMode == hipFilterModeLinear)
	{
		texDesc.normalizedCoords = true;
	}
	else
	{
		texDesc.normalizedCoords = false;
	}
	texDesc.addressMode[0] = addressMode_x;
	texDesc.addressMode[1] = addressMode_y;
	texDesc.addressMode[2] = addressMode_z;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return true;

}


bool VolumeTexture3D::initialize_array
(
	int3 dimension,
	hipTextureAddressMode addressMode_x,
	hipTextureAddressMode addressMode_y,
	hipTextureAddressMode addressMode_z
)
{

	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description
	texDesc.normalizedCoords = true;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.addressMode[0] = addressMode_x;
	texDesc.addressMode[1] = addressMode_y;
	texDesc.addressMode[2] = addressMode_z;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return true;

}


void VolumeTexture3D::release()
{
	hipFreeArray(this->cuArray_velocity);
	hipDestroyTextureObject(this->t_field);

}





bool VolumeTexture2D::initialize
(
	size_t width,
	size_t height,
	hipTextureAddressMode addressMode_x,
	hipTextureAddressMode addressMode_y
)
{


	// Allocate 2D Array
	hipChannelFormatDesc channelFormatDesc = hipCreateChannelDesc<float4>();
	gpuErrchk(hipMallocArray(&cuArray_velocity, &channelFormatDesc, width, height));


	gpuErrchk(hipMemcpy2DToArray(this->cuArray_velocity,0,0,h_field, width * sizeof(float4),width*sizeof(float4),height,hipMemcpyHostToDevice));



	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description
	texDesc.normalizedCoords = true;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.addressMode[0] = addressMode_x;
	texDesc.addressMode[1] = addressMode_y;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return true;

}


bool VolumeTexture2D::initialize_array
(
	size_t width,
	size_t height,
	hipTextureAddressMode addressMode_x,
	hipTextureAddressMode addressMode_y
)
{


	// Allocate 2D Array




	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description
	texDesc.normalizedCoords = true;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.addressMode[0] = addressMode_x;
	texDesc.addressMode[1] = addressMode_y;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return true;

}


void VolumeTexture2D::release()
{

	hipFreeArray(this->cuArray_velocity);
	hipDestroyTextureObject(this->t_field);

}