#include "hip/hip_runtime.h"
#include "HightfieldFTLE.h"
#include "..//Cuda/hip/hip_vector_types.h"
#include "..//Particle/ParticleHelperFunctions.h"
#include "DispersionHelper.h"
#include "..//Cuda/Cuda_helper_math_host.h"
#include "../Raycaster/Raycasting.h"
#include "../VolumeIO/BinaryWriter.h"


extern __constant__  BoundingBox d_boundingBox;
extern __constant__ float3 d_raycastingColor;

__host__ bool HeightfieldFTLE::InitializeParticles()
{
	this->n_particles = dispersionOptions->gridSize_2D[0] * dispersionOptions->gridSize_2D[1];
	this->h_particle = new Particle[n_particles * FTLE_NEIGHBOR] ;
	seedParticle_ZY_Plane_FTLE
	(
		h_particle,
		Array2Float3(solverOptions->gridDiameter),
		ARRAYTOINT2(dispersionOptions->gridSize_2D),
		dispersionOptions->seedWallNormalDist,
		dispersionOptions->tilt_deg,
		dispersionOptions->initial_distance
	);

	size_t Particles_byte = sizeof(Particle) * n_particles * FTLE_NEIGHBOR;

	gpuErrchk(hipMalloc((void**)&this->d_particle, Particles_byte));
	gpuErrchk(hipMemcpy(this->d_particle, this->h_particle, Particles_byte, hipMemcpyHostToDevice));

	delete[] h_particle;

	return true;
}



void HeightfieldFTLE::trace3D_path_Single()
{
	// Calculates the block and grid sizes
	unsigned int blocks;
	dim3 thread = { maxBlockDim,maxBlockDim,1 };
	blocks = BLOCK_THREAD(n_particles);

	RK4STEP RK4Step = RK4STEP::EVEN;


	// Forward FTLE
	for (int i = 0; i < solverOptions->lastIdx - solverOptions->firstIdx; i++)
	{
		if (i == 0)
		{
			// Load i 'dx field in volume_IO into field
			volume_IO.readVolume(solverOptions->currentIdx);
			// Copy and initialize velocityfield texture
			t_velocityField_0.setField(volume_IO.getField_float());
			t_velocityField_0.initialize(Array2Int3(solverOptions->gridSize),false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
			// Release the velocityfield from host (volume_IO)
			volume_IO.release();


			// Same procedure for the second texture
			volume_IO.readVolume(solverOptions->currentIdx + 1);
			t_velocityField_1.setField(volume_IO.getField_float());
			t_velocityField_1.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
			// Release the velocityfield from host (volume_IO)
			volume_IO.release();

		}
		else
		{
			// Even integration steps
			if (i % 2 == 0)
			{

				// Same procedure for the second texture
				volume_IO.readVolume(i + solverOptions->currentIdx + 1);
				t_velocityField_1.release();
				t_velocityField_1.setField(volume_IO.getField_float());
				t_velocityField_1.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
				// Release the velocityfield from host (volume_IO)
				volume_IO.release();

				RK4Step = RK4STEP::EVEN;
			}
			// Odd integration steps
			else
			{
				// Same procedure for the second texture
				volume_IO.readVolume(i + solverOptions->currentIdx + 1);
				t_velocityField_0.release();
				t_velocityField_0.setField(volume_IO.getField_float());
				t_velocityField_0.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
				// Release the velocityfield from host (volume_IO)
				volume_IO.release();

				RK4Step = RK4STEP::ODD;

			}

		}

		traceDispersion3D_path_FTLE << < blocks, thread >> >
			(
				d_particle,
				s_HeightSurface_Primary.getSurfaceObject(),
				s_HeightSurface_Primary_Extra.getSurfaceObject(),
				this->t_velocityField_0.getTexture(),
				this->t_velocityField_1.getTexture(),
				*solverOptions,
				*dispersionOptions,
				RK4Step,
				i
				);
	}


	//// Backward FTLE Calculations
	//for (int i = 0; i < solverOptions->lastIdx - solverOptions->firstIdx; i++)
	//{
	//	if (i == 0)
	//	{
	//		// Load i 'dx field in volume_IO into field
	//		volume_IO.readVolume(solverOptions->currentIdx);
	//		// Copy and initialize velocityfield texture
	//		t_velocityField_0.setField(volume_IO.getField_float());
	//		t_velocityField_0.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
	//		// Release the velocityfield from host (volume_IO)
	//		volume_IO.release();


	//		// Same procedure for the second texture
	//		volume_IO.readVolume(solverOptions->currentIdx - 1);
	//		t_velocityField_1.setField(volume_IO.getField_float());
	//		t_velocityField_1.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
	//		// Release the velocityfield from host (volume_IO)
	//		volume_IO.release();

	//	}
	//	else
	//	{
	//		// Even integration steps
	//		if (i % 2 == 0)
	//		{

	//			// Same procedure for the second texture
	//			volume_IO.readVolume(solverOptions->currentIdx - 1 - i);
	//			t_velocityField_1.release();
	//			t_velocityField_1.setField(volume_IO.getField_float());
	//			t_velocityField_1.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
	//			// Release the velocityfield from host (volume_IO)
	//			volume_IO.release();

	//			RK4Step = RK4STEP::EVEN;
	//		}
	//		// Odd integration steps
	//		else
	//		{
	//			// Same procedure for the second texture
	//			volume_IO.readVolume( solverOptions->currentIdx - 1 - i);
	//			t_velocityField_0.release();
	//			t_velocityField_0.setField(volume_IO.getField_float());
	//			t_velocityField_0.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
	//			// Release the velocityfield from host (volume_IO)
	//			volume_IO.release();

	//			RK4Step = RK4STEP::ODD;

	//		}

	//	}

	//	traceDispersion3D_path_FTLE << < blocks, thread >> >
	//		(
	//			d_particle,
	//			s_HeightSurface_Primary.getSurfaceObject(),
	//			s_HeightSurface_Primary_Extra.getSurfaceObject(),
	//			this->t_velocityField_0.getTexture(),
	//			this->t_velocityField_1.getTexture(),
	//			*solverOptions,
	//			*dispersionOptions,
	//			RK4Step,
	//			i,
	//			FTLE_Direction::BACKWARD_FTLE
	//			);
	//}




	// Calculates the gradients and store it in the cuda surface
	hipFree(d_particle);
}


void HeightfieldFTLE::rendering()
{
	this->deviceContext->PSSetSamplers(0, 1, this->samplerState.GetAddressOf());
	//this->deviceContext->OMSetBlendState(this->blendState.Get(), NULL, 0xFFFFFFFF);

	this->deviceContext->ClearRenderTargetView(this->renderTargetView.Get(), renderingOptions->bgColor);// Clear the target view

	// Calculates the block and grid sizes
	unsigned int blocks;
	dim3 thread = { maxBlockDim,maxBlockDim,1 };
	blocks = static_cast<unsigned int>((this->rays % (thread.x * thread.y) == 0 ? rays / (thread.x * thread.y) : rays / (thread.x * thread.y) + 1));



	// Depending on the Rendering mode choose the terrain Rendering function
	if (!dispersionOptions->ftleIsosurface)
	{
		if (dispersionOptions->renderingMode == dispersionOptionsMode::HeightfieldRenderingMode::SINGLE_SURFACE)
		{
			CudaTerrainRenderer_Marching_extra_FSLE << < blocks, thread >> >
				(
					this->raycastingSurface.getSurfaceObject(),
					this->volumeTexture3D_height.getTexture(),
					this->volumeTexture3D_height_extra.getTexture(),
					int(this->rays),
					this->raycastingOptions->samplingRate_0,
					this->raycastingOptions->tolerance_0,
					*dispersionOptions,
					solverOptions->lastIdx - solverOptions->firstIdx + 1
					);
		}
	}
	else
	{
		CudaRaycasting_FTLE << < blocks, thread >> >
			(
				this->raycastingSurface.getSurfaceObject(),
				this->volumeTexture3D_height.getTexture(),
				this->volumeTexture3D_height_extra.getTexture(),
				int(this->rays),
				this->raycastingOptions->samplingRate_0,
				this->raycastingOptions->tolerance_0,
				*dispersionOptions,
				solverOptions->lastIdx - solverOptions->firstIdx + 1
				);
	}

}

bool HeightfieldFTLE::singleSurfaceInitialization()
{
	
	// initialize volume Input Output
	volume_IO.Initialize(this->solverOptions);


	// Initialize Height Field as an empty cuda array 3D
	if (!this->InitializeHeightArray3D_Single
	(
		dispersionOptions->gridSize_2D[0],
		dispersionOptions->gridSize_2D[1],
		solverOptions->lastIdx - solverOptions->firstIdx
	))
		return false;



	// Bind the array of heights to the cuda surface
	if (!this->InitializeHeightSurface3D_Single())
		return false;


	// Trace particle and store their heights on the Height Surface
	this->trace3D_path_Single();


	this->gradient3D_Single_ftle();


	this->s_HeightSurface_Primary.destroySurface();
	this->s_HeightSurface_Primary_Extra.destroySurface();
		

	this->volumeTexture3D_height.setArray(a_HeightSurface_Primary.getArrayRef());
	this->volumeTexture3D_height_extra.setArray(a_HeightSurface_Primary_Extra.getArrayRef());

	this->volumeTexture3D_height.initialize_array(false,hipAddressModeClamp, hipAddressModeClamp, hipAddressModeClamp);
	this->volumeTexture3D_height_extra.initialize_array(false,hipAddressModeClamp, hipAddressModeClamp, hipAddressModeClamp);

	// calculate the correlation between ftle and height
	this->correlation();
	// Store gradient and height on the surface

	return true;		
}


void HeightfieldFTLE::gradient3D_Single_ftle()
{
	// Calculates the block and grid sizes
	unsigned int blocks;
	dim3 thread = { maxBlockDim,maxBlockDim,1 };
	blocks = static_cast<unsigned int>((this->n_particles % (thread.x * thread.y) == 0 ?
		n_particles / (thread.x * thread.y) : n_particles / (thread.x * thread.y) + 1));

	heightFieldGradient3D<FetchTextureSurface::Channel_X> << < blocks, thread >> >
		(
			s_HeightSurface_Primary.getSurfaceObject(),
			*dispersionOptions,
			*solverOptions
			);
}

void HeightfieldFTLE::correlation()
{
	//// Calculates the block and grid sizes
	//unsigned int blocks;
	//dim3 thread = { maxBlockDim,maxBlockDim,1 };
	//blocks = BLOCK_THREAD(n_particles);
	//
	//// Allocate device memory
	//gpuErrchk(hipMalloc((void**)&d_mean_ftle, sizeof(float) * solverOptions->timeSteps));
	//gpuErrchk(hipMalloc((void**)&d_mean_height, sizeof(float) * solverOptions->timeSteps));
	//gpuErrchk(hipMalloc((void**)&d_pearson_cov, sizeof(float) * solverOptions->timeSteps));
	//gpuErrchk(hipMalloc((void**)&d_pearson_var_ftle, sizeof(float) * solverOptions->timeSteps));
	//gpuErrchk(hipMalloc((void**)&d_pearson_var_height, sizeof(float) * solverOptions->timeSteps));



	//// Initialize the mean value to zero at device
	//gpuErrchk(hipMemset(this->d_mean_ftle, 0, sizeof(float) * solverOptions->timeSteps));
	//gpuErrchk(hipMemset(this->d_mean_height, 0, sizeof(float) * solverOptions->timeSteps));
	//gpuErrchk(hipMemset(this->d_pearson_cov, 0, sizeof(float) * solverOptions->timeSteps));
	//gpuErrchk(hipMemset(this->d_pearson_var_ftle, 0, sizeof(float) * solverOptions->timeSteps));
	//gpuErrchk(hipMemset(this->d_pearson_var_height, 0, sizeof(float) * solverOptions->timeSteps));




	gpuErrchk(hipMalloc((void**)&d_ftle, sizeof(float) * dispersionOptions->gridSize_2D[0]));
	gpuErrchk(hipMalloc((void**)&d_height, sizeof(float) * dispersionOptions->gridSize_2D[0]));

	fetch_ftle_height << < 1, dispersionOptions->gridSize_2D[0] >> >
		(
			volumeTexture3D_height.getTexture(),
			volumeTexture3D_height_extra.getTexture(),
			d_height,
			d_ftle,
			*solverOptions
			);

	BinaryWriter binaryWriter;
	binaryWriter.setFileName("ftleValues.bin");
	binaryWriter.setFilePath("D:\\FTLE_HEIGHT\\");
	binaryWriter.setBufferSize(sizeof(float)*dispersionOptions->gridSize_2D[0]);

	h_ftle = new float[dispersionOptions->gridSize_2D[0]];
	h_height = new float[dispersionOptions->gridSize_2D[0]];
	gpuErrchk(hipMemcpy(h_ftle, d_ftle, sizeof(float)*dispersionOptions->gridSize_2D[0], hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_height, d_height, sizeof(float)*dispersionOptions->gridSize_2D[0], hipMemcpyDeviceToHost));

	binaryWriter.setBuffer(reinterpret_cast<char*>(h_ftle));
	binaryWriter.write();
	binaryWriter.setFileName("heightValues.bin");
	binaryWriter.setBuffer(reinterpret_cast<char*>(h_height));
	binaryWriter.write();


	


	//// calculate the mean values
	//textureMean << < blocks, thread >> >
	//	(
	//		volumeTexture3D_height.getTexture(),
	//		volumeTexture3D_height_extra.getTexture(),
	//		d_mean_height,
	//		d_mean_ftle,
	//		*dispersionOptions,
	//		*solverOptions
	//	);

	//pearson_terms << < blocks, thread >> >
	//	(
	//		volumeTexture3D_height.getTexture(),
	//		volumeTexture3D_height_extra.getTexture(),
	//		d_mean_height,
	//		d_mean_ftle,
	//		d_pearson_cov,
	//		d_pearson_var_ftle,
	//		d_pearson_var_height,
	//		*dispersionOptions,
	//		*solverOptions
	//		);

	//fetchftle_height << < blocks, thread >> >
	//(
	//	volumeTexture3D_height.getTexture(),
	//	volumeTexture3D_height_extra.getTexture(),
	//	d_mean_height,
	//	d_mean_ftle,
	//	*dispersionOptions,
	//	*solverOptions
	//);

	//pearson << < 1, solverOptions->timeSteps >> >
	//	(
	//		d_pearson_cov,
	//		d_pearson_var_ftle,
	//		d_pearson_var_height,
	//		*solverOptions
	//		);

	//h_pearson = new float[solverOptions->timeSteps];
	//gpuErrchk(hipMemcpy(h_pearson, d_pearson_cov, sizeof(float)*solverOptions->timeSteps, hipMemcpyDeviceToHost));

	//


	//gpuErrchk(hipFree(d_mean_ftle));
	//gpuErrchk(hipFree(d_mean_height));
	//gpuErrchk(hipFree(d_pearson_var_ftle));
	//gpuErrchk(hipFree(d_pearson_var_height));

}


__host__ bool HeightfieldFTLE::initializeBoundingBox()
{

	BoundingBox* h_boundingBox = new BoundingBox;


	h_boundingBox->gridSize = make_int3(dispersionOptions->gridSize_2D[0], dispersionOptions->gridSize_2D[1], solverOptions->lastIdx - solverOptions->firstIdx + 1);
	h_boundingBox->updateBoxFaces(ArrayFloat3ToFloat3(solverOptions->gridDiameter));
	h_boundingBox->updateAspectRatio(*width, *height);

	h_boundingBox->constructEyeCoordinates
	(
		XMFloat3ToFloat3(camera->GetPositionFloat3()),
		XMFloat3ToFloat3(camera->GetViewVector()),
		XMFloat3ToFloat3(camera->GetUpVector())
	);

	h_boundingBox->FOV = (this->FOV_deg / 360.0f) * XM_2PI;
	h_boundingBox->distImagePlane = this->distImagePlane;

	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_boundingBox), h_boundingBox, sizeof(BoundingBox)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_raycastingColor), this->raycastingOptions->color_0, sizeof(float3)));


	delete h_boundingBox;

	return true;
}