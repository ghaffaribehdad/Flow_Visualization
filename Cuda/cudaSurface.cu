#include "cudaSurface.h"

bool CudaSurface::initializeSurface()
{
	// Specify surface
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;

	// Create the surface objects
	resDesc.res.array.array = this->cuInputArray;

	hipCreateSurfaceObject(&surfaceObject, &resDesc);

	return true;
}


bool CudaSurface::destroySurface()
{

	// Destroy surface objects
	hipDestroySurfaceObject(surfaceObject);

	

	return true;
}
