#include "hip/hip_runtime.h"
#include "StreamlineSolver.cuh"

// Explicit instantions
template class StreamlineSolver<float>;
template class StreamlineSolver<double>;


template <typename T>
__host__ bool StreamlineSolver<T>::solve()
{
	this->volume_IO.Initialize(this->solverOptions);

	// TO-DO: Define streamlinesolver for double precision
	InitializeVelocityField();

	this->InitializeTexture();

	this->InitializeParticles();

	TracingParticles<T> << <1, solverOptions.lines_count >> > (d_Particles, t_VelocityField, solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer));
	//TracingParticles<T> << < 1, solverOptions.particle_count >> > (d_Particles, t_VelocityField, solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer));

	hipFree(d_Particles);
	hipFree(d_VelocityField);
	return true;
}

template<typename T>
__host__ void StreamlineSolver<T>::InitializeVelocityField()
{
	this->volume_IO.readVolume(solverOptions.currentIdx);
	std::vector<char>* p_vec_buffer = volume_IO.flushBuffer();
	char* p_vec_buffer_temp = &(p_vec_buffer->at(0));

	this->h_VelocityField = reinterpret_cast<T*> (p_vec_buffer_temp);
}

template <typename T>
void StreamlineSolver<T>::InitializeParticles()
{
	// Create an array of particles
	this->h_Particles = new Particle<T>[solverOptions.lines_count];

	float3 gridDiameter =
	{
		solverOptions.gridDiameter[0],
		solverOptions.gridDiameter[1],
		solverOptions.gridDiameter[2]
	};
	// Seed Particles Randomly according to the grid diameters
	for (int i = 0; i < solverOptions.lines_count; i++)
	{
		h_Particles[i].seedParticle(gridDiameter);
	}

	size_t Particles_byte = sizeof(*h_Particles) * solverOptions.lines_count;

	// Upload Velocity Filled to GPU 

	gpuErrchk(hipMalloc((void**)& d_Particles, Particles_byte));

	gpuErrchk(hipMemcpy(d_Particles, h_Particles, Particles_byte, hipMemcpyHostToDevice));

	delete h_Particles;
}


template <typename T>
__host__ bool StreamlineSolver<T>::InitializeTexture()
{

	// Cuda 3D array of velocities
	hipArray_t cuArray_velocity;


	// define the size of the velocity field
	hipExtent extent =
	{
		static_cast<size_t>(solverOptions.gridSize[0]),
		static_cast<size_t>(solverOptions.gridSize[1]),
		static_cast<size_t>(solverOptions.gridSize[2])
	};


	// Allocate 3D Array
	hipChannelFormatDesc channelFormatDesc = hipCreateChannelDesc<float4>();
	gpuErrchk(hipMalloc3DArray(&cuArray_velocity, &channelFormatDesc, extent, 0));



	// set copy parameters to copy from velocity field to array
	hipMemcpy3DParms cpyParams = { 0 };

	cpyParams.srcPtr = make_hipPitchedPtr((void*)this->h_VelocityField, extent.width * sizeof(float4), extent.height, extent.depth);
	cpyParams.dstArray = cuArray_velocity;
	cpyParams.kind = hipMemcpyHostToDevice;
	cpyParams.extent = extent;


	// Copy velocities to 3D Array
	gpuErrchk(hipMemcpy3D(&cpyParams));
	// might need sync before release the host memory

	// Release the Volume while it is copied on GPU
	this->volume_IO.release();


	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray_velocity;

	// Texture Description
	texDesc.normalizedCoords = true;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.addressMode[2] = hipAddressModeClamp;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_VelocityField, &resDesc, &texDesc, NULL));

	hipDestroyTextureObject(t_VelocityField);
	return true;

}