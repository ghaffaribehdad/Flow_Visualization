#include "hip/hip_runtime.h"
#include "StreamlineSolver.cuh"


// Explicit instantions
template class StreamlineSolver<float>;
template class StreamlineSolver<double>;



template<typename T>
__host__ T* StreamlineSolver<T>::InitializeVelocityField()
{
	std::vector<char>* p_vec_buffer = this->volume_IO.readVolume(0);

	char* p_vec_buffer_temp = &(p_vec_buffer->at(0));

	this->h_VelocityField = reinterpret_cast<T*> (p_vec_buffer_temp);

	size_t velocityField_byte =
		solverOptions.gridSize[0] *
		solverOptions.gridSize[1] *
		solverOptions.gridSize[2] * 3 * sizeof(T);

	// Allocate memory on Device
	gpuErrchk(hipMalloc((void**)& d_VelocityField, velocityField_byte));

	// Upload Velocity Filled to GPU 
	gpuErrchk(hipMemcpy(d_VelocityField, h_VelocityField, velocityField_byte, hipMemcpyHostToDevice));

	return h_VelocityField;
}

template <typename T>
void StreamlineSolver<T>::InitializeParticles()
{
	// Create an array of particles
	this->h_Particles = new Particle<T>[solverOptions.particle_count];

	float3 gridDiameter =
	{
		solverOptions.gridDiameter[0],
		solverOptions.gridDiameter[1],
		solverOptions.gridDiameter[2]
	};
	// Seed Particles Randomly according to the grid diameters
	for (int i = 0; i < solverOptions.particle_count; i++)
	{
		h_Particles[i].seedParticle(gridDiameter);
	}
	
	size_t Particles_byte = sizeof(*h_Particles) * solverOptions.particle_count;

	// Upload Velocity Filled to GPU 

	gpuErrchk(hipMalloc((void**)& d_Particles, Particles_byte));

	gpuErrchk(hipMemcpy(d_Particles, h_Particles, Particles_byte, hipMemcpyHostToDevice));

	delete h_Particles;
}


template <typename T>
__host__ bool StreamlineSolver<T>::solve()
{
	float * h_VelocityField = InitializeVelocityField();
	this->InitializeTexture(h_VelocityField);
	this->InitializeParticles();

	TracingParticles<T> << < 1, solverOptions.particle_count >> > (d_Particles, d_VelocityField, solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer));
	//TracingParticles<T> << < 1, solverOptions.particle_count >> > (d_Particles, t_VelocityField, solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer));

	hipFree(d_Particles);
	hipFree(d_VelocityField);
	return true;
}


template <typename T>
__host__ bool StreamlineSolver<T>::InitializeTexture(T* h_VelocityField)
{

	// Cuda 3D array of velocities
	hipArray_t cuArray_velocity;


	// define the size of the velocity field
	hipExtent extent = 
	{ 
		solverOptions.gridSize[0],
		solverOptions.gridSize[1],
		solverOptions.gridSize[2] 
	};


	// Allocate 3D Array
	hipChannelFormatDesc channelFormatDesc = hipCreateChannelDesc<float4>();
	gpuErrchk(hipMalloc3DArray(&cuArray_velocity, &channelFormatDesc, extent, 0));



	// set copy parameters to copy from velocity field to array
	hipMemcpy3DParms cpyParams = { 0 };

	cpyParams.srcPtr = make_hipPitchedPtr((void*)h_VelocityField, extent.width * sizeof(float4), extent.height, extent.depth);
	cpyParams.dstArray = cuArray_velocity;
	cpyParams.kind = hipMemcpyHostToDevice;
	cpyParams.extent = extent;


	// Copy velocities to 3D Array
	gpuErrchk(hipMemcpy3D(&cpyParams));



	//// Create a texture
	hipTextureObject_t t_VelocityField = 0;

	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray_velocity;

	// Texture Description
	texDesc.normalizedCoords = true;
	texDesc.filterMode = cudaFilterModeLi;
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.addressMode[2] = hipAddressModeClamp;
	texDesc.readMode = hipReadModeElementType;



	gpuErrchk(hipCreateTextureObject(&t_VelocityField, &resDesc, &texDesc, NULL));

	//hipDestroyTextureObject(t_VelocityField);
	return true;

}