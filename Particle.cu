#include "hip/hip_runtime.h"
#include "Particle.cuh"

// Explicit instantions
template class Particle<float>;
template class Particle<double>;

template <typename T>
__device__  void Particle<T>::updateVelocity(const float3& gridDiameter, const int3& gridSize, hipTextureObject_t t_VelocityField)
{
	if (!outOfScope)
	{
		float3 linIndex = findRelative(gridDiameter);
		float4 velocity4D = tex3D<float4>(t_VelocityField, linIndex.x, linIndex.y, linIndex.z);
		float3 velocity = { velocity4D.x,velocity4D.y,velocity4D.z };
		this->setVelocity(velocity);
	}

}

template <typename T>
__device__ __host__ float3 Particle<T>::findRelative(const float3& gridDiameter)
{
	float3 relative_position = {
		(static_cast<float>(this->m_position.x)) / (gridDiameter.x),
		(static_cast<float>(this->m_position.y)) / (gridDiameter.y),
		(static_cast<float>(this->m_position.z)) / (gridDiameter.z)
	};
	return relative_position;
}
