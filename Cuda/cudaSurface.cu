#include "cudaSurface.cuh"

bool CudaSurface::initializeSurface()
{


	// Allocate CUDA arrays in device memory
	//hipChannelFormatDesc channelDesc =
	//	hipCreateChannelDesc(8, 8, 8, 8,
	//		hipChannelFormatKindUnsigned);


	//hipMallocArray(&this->cuInputArray, &channelDesc, this->width, this->height,\
		//hipArraySurfaceLoadStore);



	// Specify surface
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;

	// Create the surface objects
	resDesc.res.array.array = this->cuInputArray;

	hipCreateSurfaceObject(&surfaceObject, &resDesc);

	return true;
}


bool CudaSurface::destroySurface()
{
	// Free device memory
	//hipFreeArray(cuInputArray);

	// Destroy surface objects
	hipDestroySurfaceObject(surfaceObject);

	

	return true;
}
