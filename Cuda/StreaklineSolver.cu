#include "StreaklineSolver.h"
#include "CudaHelperFunctions.h"



void StreaklineSolver::release()
{
	hipFree(this->d_Particles);
	hipFree(this->d_VelocityField);


	this->volumeTexture_0.release();
	this->volumeTexture_1.release();
}

__host__ bool StreaklineSolver::initializeRealtime()
{
	//At least two timesteps is needed
	this->timeSteps = solverOptions->lastIdx - solverOptions->firstIdx;

	// Initialize Volume IO (Save file path and file names)
	this->volume_IO.InitializeRealTime(this->solverOptions);

	// Initialize Particles and upload it to GPU
	this->InitializeParticles(this->solverOptions->seedingPattern);

	int blockDim = 256;
	int thread = (this->solverOptions->lines_count / blockDim) + 1;

	// set the position of the vertex buffer to the intial position of the particle
	InitializeVertexBufferStreaklines << <blockDim, thread >> >
		(this->d_Particles,
			*solverOptions,
			reinterpret_cast<Vertex*>(this->p_VertexBuffer)
			);

	solverOptions->lineLength = timeSteps;

	return true;
}

__host__ bool StreaklineSolver::solve()
{
	//At least two timesteps is needed
	int timeSteps = solverOptions->lastIdx - solverOptions->currentIdx;

	// Initialize Volume IO (Save file path and file names)
	this->volume_IO.Initialize(this->solverOptions);

	// Initialize Particles and upload it to GPU
	this->InitializeParticles(solverOptions->seedingPattern);

	// Number of threads based on the number of lines
	int blockDim = 256;
	int thread = (this->solverOptions->lines_count / blockDim) + 1;

	solverOptions->lineLength = timeSteps;
	bool odd = false;

	// set the position of the vertex buffer to the intial position of the particle
	InitializeVertexBufferStreaklines << <blockDim, thread >> >
		(	this->d_Particles,
			*solverOptions,
			reinterpret_cast<Vertex*>(this->p_VertexBuffer)
			);

	// we go through each time step and solve RK4 for even time steps the first texture is updated,
	// while the second texture is updated for odd time steps
	for (int step = 0; step < timeSteps; step++)
	{
		// First Step
		if (step == 0)
		{

			// Read current volume
			this->volume_IO.readVolume(solverOptions->currentIdx);
			// Return a pointer to volume
			this->h_VelocityField = this->volume_IO.getField_float();
			// set the pointer to the volume texture
			this->volumeTexture_0.setField(h_VelocityField);
			// initialize the volume texture
			this->volumeTexture_0.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);
			// release host memory
			volume_IO.release();



			// same procedure for the second field
			this->volume_IO.readVolume(solverOptions->currentIdx + 1);
			this->h_VelocityField = this->volume_IO.getField_float();
			this->volumeTexture_1.setField(h_VelocityField);
			this->volumeTexture_1.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);

			volume_IO.release();

		}

		else if (step % 2 == 0) // => EVEN
		{
			this->volume_IO.readVolume(solverOptions->currentIdx + step + 1);
			this->h_VelocityField = this->volume_IO.getField_float();

			this->volumeTexture_1.release();
			this->volumeTexture_1.setField(h_VelocityField);
			this->volumeTexture_1.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);

			volume_IO.release();


			odd = false;
		}

		else if (step % 2 != 0) // => ODD
		{

			this->volume_IO.readVolume(solverOptions->currentIdx + step + 1);
			this->h_VelocityField = this->volume_IO.getField_float();

			this->volumeTexture_0.release();
			this->volumeTexture_0.setField(h_VelocityField);
			this->volumeTexture_0.initialize(Array2Int3(solverOptions->gridSize), false, hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap);

			volume_IO.release();

			odd = true;

		}

		TracingStreak << <blockDim, thread >> >
			(
				volumeTexture_0.getTexture(),
				volumeTexture_1.getTexture(),
				*solverOptions,
				reinterpret_cast<Vertex*>(this->p_VertexBuffer),
				odd,
				step
				);



	}

	// Bring the position to the middle
	AddOffsetVertexBufferStreaklines << <blockDim, thread >> >(*solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer));
	this->release();
	return true;
}


__host__ bool StreaklineSolver::solveRealtime()
{
	int blockDim = 256;
	int thread = (this->solverOptions->lines_count / blockDim) + 1;

	bool odd = false;


	// First Step

	switch (solverOptions->Compressed)
	{

	case true: // Compressed Data
	{
		if (solverOptions->counter == 0)
		{
			initializeTextureCompressed(solverOptions, volumeTexture_0, solverOptions->currentIdx);
			initializeTextureCompressed(solverOptions, volumeTexture_1, solverOptions->currentIdx + 1);
		}
		else if (this->solverOptions->counter % 2 == 0) // => EVEN
		{
			this->volumeTexture_1.release();
			initializeTextureCompressed(solverOptions, volumeTexture_1, solverOptions->currentIdx + solverOptions->counter + 1);
			odd = false;
		}
		else if (this->solverOptions->counter % 2 != 0) // => ODD
		{
			this->volumeTexture_0.release();
			initializeTextureCompressed(solverOptions, volumeTexture_0, solverOptions->currentIdx + solverOptions->counter + 1);
			odd = true;

		}

		break;
	}

	case false: // Uncompressed Data
	{
		if (solverOptions->counter == 0)
		{
			initializeTexture(solverOptions, volumeTexture_0, solverOptions->currentIdx);
			initializeTexture(solverOptions, volumeTexture_1, solverOptions->currentIdx + 1);
		}
		else if (solverOptions->counter % 2 == 0) // => EVEN
		{
			this->volumeTexture_1.release();
			initializeTexture(solverOptions, volumeTexture_1, solverOptions->currentIdx + solverOptions->counter + 1);
			odd = false;

		}
		else if (solverOptions->counter % 2 != 0) // => ODD
		{
			this->volumeTexture_0.release();
			initializeTexture(solverOptions, volumeTexture_0, solverOptions->currentIdx + solverOptions->counter + 1);
			odd = true;
		}
		break;
	}

	}

	//int numofBlock = 0;
	//hipOccupancyMaxActiveBlocksPerMultiprocessor(&numofBlock, TracingPath, blockDim, 0);
	//std::printf("Optimized number of blocks are  %d \n", numofBlock);	

	TracingStreak << <blockDim, thread >> > (volumeTexture_0.getTexture(), volumeTexture_1.getTexture(), *solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer), odd, solverOptions->counter);
	std::printf("\n\n");

	if (solverOptions->counter == timeSteps)
	{
		solverOptions->drawComplete = true;
		this->release();
	}
	else
	{
		solverOptions->counter++;
	}


	return true;
}



