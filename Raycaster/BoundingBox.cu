#include "hip/hip_runtime.h"
#include "BoundingBox.h"
#include <iostream>
#include <fstream>
#include "../Cuda/hip/hip_vector_types.h"

__device__ __host__ void BoundingBox::constructEyeCoordinates(const float3& eyePos, const float3& viewDir, const float3& upVec)
{

	this->m_eyePos = eyePos;
	this->m_viewDir = viewDir;
	this->m_upVec = upVec;

	// N vector
	this->nuv[0] = normalize(this->m_viewDir);

	// U vector
	float3 upVectNorm = normalize(this->m_upVec);
	this->nuv[1] = cross(nuv[0], upVectNorm);

	// V vector
	this->nuv[2] = cross(nuv[0], nuv[1]);
}


__host__ __device__ void BoundingBox::updateBoxFaces(const float3 & dimensions)
{
	m_dimensions = dimensions;
	this->boxFaces[0] = dimensions.x / -2.0f;
	this->boxFaces[1] = dimensions.x / 2.0f;

	this->boxFaces[2] = dimensions.y / -2.0f;
	this->boxFaces[3] = dimensions.y / 2.0f;

	this->boxFaces[4] = dimensions.z / -2.0f;
	this->boxFaces[5] = dimensions.z / 2.0f;


}

__host__ __device__ void BoundingBox::updateAspectRatio(const int & width, const int & height)
{
	this->m_width = width;
	this->m_height = height;
	this->aspectRatio = static_cast<float>(this->m_width) / static_cast<float>(this->m_height);
}