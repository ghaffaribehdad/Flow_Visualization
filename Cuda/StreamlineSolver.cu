#include "hip/hip_runtime.h"
#include "StreamlineSolver.h"
#include "hip/hip_vector_types.h"
#include "..//Cuda/CudaHelperFunctions.h"
#include ""


// Kernel of the streamlines, TO-DO: Divide kernel into seprate functions

__global__ void TracingStream(Particle* d_particles, hipTextureObject_t t_VelocityField, SolverOptions solverOptions, Vertex* p_VertexBuffer)
{
	unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

	if (index < solverOptions.lines_count)
	{
		int lineLength = solverOptions.lineLength;
		int index_buffer = index * lineLength;
		float dt = solverOptions.dt;
		float3 gridDiameter =
		{
			solverOptions.gridDiameter[0],
			solverOptions.gridDiameter[1],
			solverOptions.gridDiameter[2]
		};

		float3 temp_position = *d_particles[index].getPosition();

		d_particles[index].updateVelocity(gridDiameter, t_VelocityField);
		

		float3 upDir = make_float3(0.0f, 1.0f, 0.0f);

		if (abs(dot(upDir, normalize(d_particles[index].m_velocity))) > 0.1f)
			upDir = make_float3(1.0f, 0.0f, 0.0f);

		if (abs(dot(upDir, normalize(d_particles[index].m_velocity))) > 0.1f)
			upDir = make_float3(0.0f, 0.0f, 1.0f);



		for (int i = 0; i < lineLength; i++)
		{
			if (solverOptions.periodic)
			{
				p_VertexBuffer[index_buffer + i].pos.x = d_particles[index].getPosition()->x - (gridDiameter.x / 2.0);
				p_VertexBuffer[index_buffer + i].pos.y = d_particles[index].getPosition()->y - (gridDiameter.y / 2.0);
				p_VertexBuffer[index_buffer + i].pos.z = d_particles[index].getPosition()->z - (gridDiameter.z / 2.0);
			}
			else 
			{
				if (!d_particles[index].isOut())
				{
					d_particles[index].checkPosition(gridDiameter);
				}

				if (d_particles[index].isOut() && i != 0)
				{
					p_VertexBuffer[index_buffer + i].pos.x = p_VertexBuffer[index_buffer + i - 1].pos.x;
					p_VertexBuffer[index_buffer + i].pos.y = p_VertexBuffer[index_buffer + i - 1].pos.y;
					p_VertexBuffer[index_buffer + i].pos.z = p_VertexBuffer[index_buffer + i - 1].pos.z;
				}
				else
				{
					p_VertexBuffer[index_buffer + i].pos.x = d_particles[index].getPosition()->x - (gridDiameter.x / 2.0);
					p_VertexBuffer[index_buffer + i].pos.y = d_particles[index].getPosition()->y - (gridDiameter.y / 2.0);
					p_VertexBuffer[index_buffer + i].pos.z = d_particles[index].getPosition()->z - (gridDiameter.z / 2.0);

				}
			}



			float3* velocity = d_particles[index].getVelocity();
			float3 tangent = normalize(*velocity);
			


			p_VertexBuffer[index_buffer + i].normal.x = upDir.x;
			p_VertexBuffer[index_buffer + i].normal.y = upDir.y;
			p_VertexBuffer[index_buffer + i].normal.z = upDir.z;

			p_VertexBuffer[index_buffer + i].tangent.x = tangent.x;
			p_VertexBuffer[index_buffer + i].tangent.y = tangent.y;
			p_VertexBuffer[index_buffer + i].tangent.z = tangent.z;


			
			p_VertexBuffer[index_buffer + i].LineID = index;
			

			switch (solverOptions.colorMode)
			{
				case 0: // Velocity
				{

					p_VertexBuffer[index_buffer + i].measure = VecMagnitude(*velocity);
					break;

				}
				case 1: // Vx
				{
					p_VertexBuffer[index_buffer + i].measure = d_particles[index].getVelocity()->x;;
					break;
				}
				case 2: // Vy
				{
					p_VertexBuffer[index_buffer + i].measure = d_particles[index].getVelocity()->y;
					break;
				}
				case 3: // Vz
				{
					p_VertexBuffer[index_buffer + i].measure = d_particles[index].getVelocity()->z;
					break;
				}
			}

			// Do not check if it is out
			RK4Stream(t_VelocityField, &d_particles[index], gridDiameter, dt);

			// Update position based on the projection
			switch (solverOptions.projection)
			{
				case Projection::NO_PROJECTION:
				{
					break;
				}
				case Projection::ZY_PROJECTION:
				{
					p_VertexBuffer[index_buffer + i].pos.x = temp_position.x - (gridDiameter.x / 2.0);
	

					break;
				}
				case Projection::XZ_PROJECTION:
				{
					p_VertexBuffer[index_buffer + i].pos.y = temp_position.y - (gridDiameter.y / 2.0);


					break;
				}
				case Projection::XY_PROJECTION:
				{

					p_VertexBuffer[index_buffer + i].pos.z = temp_position.z - (gridDiameter.z / 2.0);

					break;
				}
			}




		}//end of for loop
	}
}

__host__ void StreamlineSolver::release()
{
	hipFree(this->d_Particles);
	hipFree(this->d_VelocityField);
	this->volumeTexture.release();
}

__host__ bool StreamlineSolver::solve()
{
	// Read Dataset
	this->volume_IO.Initialize(this->solverOptions);
	this->h_VelocityField = InitializeVelocityField(this->solverOptions->currentIdx);
	
	// Copy data to the texture memory
	this->volumeTexture.setField(h_VelocityField);
	this->volumeTexture.setSolverOptions(this->solverOptions);
	this->volumeTexture.initialize();


	// Release it from Host
	volume_IO.release();
	

	this->InitializeParticles(static_cast<SeedingPattern>( this->solverOptions->seedingPattern));

	int blockDim = 1024;
	int thread = (this->solverOptions->lines_count / blockDim)+1;
	
	TracingStream << <blockDim , thread >> > (this->d_Particles, volumeTexture.getTexture(), *this->solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer));

	this->release();

	return true;
}