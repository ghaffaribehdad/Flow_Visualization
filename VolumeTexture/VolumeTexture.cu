#include "hip/hip_runtime.h"

#include "VolumeTexture.h"
#include "..//ErrorLogger.h"

void VolumeTexture::setGridDiameter(const float3& _gridDiamter)
{
	this->gridDiameter = _gridDiamter;
}

void VolumeTexture::setGridSize(const int3& _gridSize)
{
	this->gridSize = _gridSize;
}

void VolumeTexture::setField(float* _h_field)
{
	this->h_field = _h_field;
}

void VolumeTexture::initialize()
{
	// Cuda 3D array of velocities
	hipArray_t cuArray_velocity;


	// define the size of the velocity field
	hipExtent extent =
	{
		static_cast<size_t>(this->gridSize.x),
		static_cast<size_t>(this->gridSize.y),
		static_cast<size_t>(this->gridSize.z)
	};


	// Allocate 3D Array
	hipChannelFormatDesc channelFormatDesc = hipCreateChannelDesc<float4>();
	gpuErrchk(hipMalloc3DArray(&cuArray_velocity, &channelFormatDesc, extent, 0));



	// set copy parameters to copy from velocity field to array
	hipMemcpy3DParms cpyParams = { 0 };

	cpyParams.srcPtr = make_hipPitchedPtr((void*)this->h_field, extent.width * sizeof(float4), extent.height, extent.depth);
	cpyParams.dstArray = cuArray_velocity;
	cpyParams.kind = hipMemcpyHostToDevice;
	cpyParams.extent = extent;


	// Copy velocities to 3D Array
	gpuErrchk(hipMemcpy3D(&cpyParams));
	// might need sync before release the host memory

	// Release the Volume while it is copied on GPU
	//this->volume_IO.release();


	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray_velocity;

	// Texture Description
	texDesc.normalizedCoords = true;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.addressMode[2] = hipAddressModeClamp;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

}
void VolumeTexture::release()
{
	gpuErrchk(hipDestroyTextureObject(this->t_field));
}


const int3 & VolumeTexture::getGridSize() const
{
	return this->gridSize;
}
const float3& VolumeTexture::getGridDiameter() const
{
	return this->gridDiameter;
}


__device__ float4 VolumeTexture::fetch(float3 index)
{
	return tex3D<float4>(this->t_field, index.x, index.y, index.z);
}