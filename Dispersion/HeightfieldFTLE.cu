#include "HightfieldFTLE.h"
#include "..//Cuda/hip/hip_vector_types.h"
#include "..//Particle/ParticleHelperFunctions.h"
#include "DispersionHelper.h"


__host__ bool HeightfieldFTLE::InitializeParticles()
{
	this->n_particles = dispersionOptions->gridSize_2D[0] * dispersionOptions->gridSize_2D[1] * FTLE_NEIGHBOR;
	this->h_particle = new Particle[n_particles];
	seedParticle_ZY_Plane_FTLE
	(
		h_particle,
		ARRAYTOFLOAT3(solverOptions->gridDiameter),
		ARRAYTOINT2(dispersionOptions->gridSize_2D),
		dispersionOptions->seedWallNormalDist,
		dispersionOptions->tilt_deg,
		dispersionOptions->ftleDistance
	);

	size_t Particles_byte = sizeof(Particle) * n_particles;

	gpuErrchk(hipMalloc((void**)&this->d_particle, Particles_byte));
	gpuErrchk(hipMemcpy(this->d_particle, this->h_particle, Particles_byte, hipMemcpyHostToDevice));

	delete[] h_particle;

	return true;
}



void HeightfieldFTLE::trace3D_path_Single()
{
	// Calculates the block and grid sizes
	unsigned int blocks;
	dim3 thread = { maxBlockDim,maxBlockDim,1 };
	blocks = BLOCK_THREAD(n_particles);

	RK4STEP RK4Step = RK4STEP::ODD;

	for (int i = 0; i < solverOptions->lastIdx - solverOptions->firstIdx; i++)
	{
		if (i == 0)
		{
			// Load i 'dx field in volume_IO into field
			this->LoadVelocityfield(i + solverOptions->currentIdx);
			// Copy and initialize velocityfield texture
			this->initializeVolumeTexuture(hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap, velocityField_0);
			// Release the velocityfield from host (volume_IO)
			primary_IO.release();

			// Same procedure for the second texture
			this->LoadVelocityfield(i + solverOptions->currentIdx + 1);
			this->initializeVolumeTexuture(hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap, velocityField_1);
			primary_IO.release();

		}
		else
		{
			// Even integration steps
			if (i % 2 == 0)
			{

				this->LoadVelocityfield(i + solverOptions->currentIdx);
				this->velocityField_1.release();
				this->initializeVolumeTexuture(hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap, velocityField_1);
				primary_IO.release();

				RK4Step = RK4STEP::ODD;
			}
			// Odd integration steps
			else
			{
				this->LoadVelocityfield(i + solverOptions->currentIdx);
				this->velocityField_0.release();
				this->initializeVolumeTexuture(hipAddressModeWrap, hipAddressModeBorder, hipAddressModeWrap, velocityField_0);
				primary_IO.release();

				RK4Step = RK4STEP::EVEN;

			}

		}

		// initialize proper velocityfield

		// trace
		traceDispersion3D_path << < blocks, thread >> >
			(
				d_particle,
				s_HeightSurface_Primary.getSurfaceObject(),
				s_HeightSurface_Primary_Ex.getSurfaceObject(),
				this->velocityField_0.getTexture(),
				this->velocityField_1.getTexture(),
				*solverOptions,
				*dispersionOptions,
				RK4Step,
				i
				);
	}
	// Calculates the gradients and store it in the cuda surface
	hipFree(d_particle);
}