#include "hip/hip_runtime.h"
#include "Raycasting.h"



__constant__ BoundingBox d_boundingBox;



__host__ bool Raycasting::initialize()
{
	// set the number of rays = number of pixels
	this->rays = this->raycasting_desc.width * this->raycasting_desc.height;

	// initialize the bounding box
	initilizeBoundingBox();

	// Read and set field
	if(!fileLoaded)
	{
		this->volume_IO.Initialize(this->raycasting_desc.solverOption);
		this->initializeIO();
		this->initializeVolumeTexuture();

		fileLoaded = true;
	}
	if (fileChanged)
	{
		this->initializeIO();
		this->volumeTexture.release();
		this->initializeVolumeTexuture();

		fileChanged = false;
	}

	
	return true;

}

__host__ bool Raycasting::release()
{
	//hipFree(d_boundingBox);
	//this->volumeTexture.release();

	return true;
}

__host__ void Raycasting::Rendering()
{


	int blocks = (int(rays) % (maxThreadBlock )) == 0 ? int(rays) / maxThreadBlock : (int(rays) / maxThreadBlock ) + 1;
	isoSurfaceVelocityMagnitude <<< blocks, maxThreadBlock >> > 
		(
		this->raycastingSurface->getSurfaceObject(),
		this->volumeTexture.getTexture(),
		int(this->rays),
		15.0f,
		.02f,
		.09f
	);
	//hipDeviceSynchronize();
}


__global__ void boundingBoxRendering(hipSurfaceObject_t raycastingSurface, hipTextureObject_t field1, int rays)
{
	//// Calculate surface coordinates
	//int thread = threadIdx.x;
	//
	//int index = blockIdx.x * blockDim.x + thread;


	//
	//if (index < rays)
	//{
	//	int2 pixel = { 0,0 };
	//	pixel.x = index / d_boundingBox.getResolution().y;
	//	pixel.y = index - (pixel.x * d_boundingBox.getResolution().y);
	//	float3 pixelPos = d_boundingBox.pixelPosition(pixel.x, pixel.y);
	//	float2 NearFar = d_boundingBox.findIntersections(pixelPos);

	//	if (NearFar.x != -1)
	//	{
	//		float4 color = { 1,1,0,0 };
	//		float rgba = DecodeFloatRGBA(color);
	//		surf2Dwrite(rgba, raycastingSurface, sizeof(float) * pixel.x, pixel.y);
	//	}

	//}
}

__host__ bool Raycasting::initilizeBoundingBox()
{
	BoundingBox * h_boundingBox = new BoundingBox;

	h_boundingBox->eyePos = this->raycasting_desc.eyePos;
	h_boundingBox->viewDir = this->raycasting_desc.viewDir;
	h_boundingBox->upVec = this->raycasting_desc.upDir;

	h_boundingBox->width = this->raycasting_desc.width;
	h_boundingBox->height= this->raycasting_desc.height;
	h_boundingBox->gridDiameter = this->raycasting_desc.gridDiameter;
	h_boundingBox->updateBoxFaces();
	h_boundingBox->updateAspectRatio();
	h_boundingBox->constructEyeCoordinates();
	h_boundingBox->FOV = (this->raycasting_desc.FOV_deg) * 3.1415f / 180.0f;
	h_boundingBox->distImagePlane = 1;



	// Populate the constant memory
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_boundingBox), h_boundingBox, sizeof(BoundingBox)));
	

	delete h_boundingBox;





	return true;
}


__host__ bool Raycasting::initializeVolumeTexuture()
{
	this->volumeTexture.setGridDiameter(this->raycasting_desc.gridDiameter);
	this->volumeTexture.setGridSize(this->raycasting_desc.gridSize);
	this->volumeTexture.setField(this->raycasting_desc.field);
	this->volumeTexture.initialize();

	return true;
}

__host__ bool Raycasting::initializeIO()
{
	
	this->volume_IO.readVolume(this->raycasting_desc.solverOption.currentIdx);
	std::vector<char>* p_vec_buffer = volume_IO.flushBuffer();
	char* p_vec_buffer_temp = &(p_vec_buffer->at(0));
	raycasting_desc.field = reinterpret_cast<float*>(p_vec_buffer_temp);
	
	return true;
}




__global__ void isoSurfaceVelocityMagnitude(hipSurfaceObject_t raycastingSurface, hipTextureObject_t field1, int rays, float isoValue, float samplingRate, float IsosurfaceTolerance)
{
	// Calculate surface coordinates
	int thread = threadIdx.x;

	int index = blockIdx.x * blockDim.x + thread;


	if (index < rays)
	{
		int2 pixel = { 0,0 };
		pixel.y = index / d_boundingBox.width;
		pixel.x = index - pixel.y * d_boundingBox.width;
		float3 pixelPos = pixelPosition(d_boundingBox,pixel.x, pixel.y);
		float2 NearFar = findIntersections(pixelPos, d_boundingBox);
		
		float4 velocity4D = { 0,0,0,0 };
		float4 color = {.5,.2,.5,0 };
		float rgba = rgba = DecodeFloatRGBA(color);;


		// if hits
		if (NearFar.y != -1)
		{ 
			float3 rayDir = normalize(pixelPos - d_boundingBox.eyePos);
			
			for (float t = NearFar.x; t < NearFar.y; t = t + samplingRate)
			{

				float3 relativePos = (d_boundingBox.eyePos + t * rayDir);
				relativePos += d_boundingBox.gridDiameter / 2.0;
				relativePos = relativePos/ d_boundingBox.gridDiameter;

				velocity4D = tex3D<float4>(field1, relativePos.x, relativePos.y, relativePos.z);

				if (fabsf(velocityMagnitude(velocity4D) - isoValue) < IsosurfaceTolerance)
				{
					surf2Dwrite(rgba, raycastingSurface, 4* pixel.x, pixel.y);
					break;
				}

			}

		}

	/*	if (NearFar.y != -1)
		{

			float4 color = { 1,.5,.5,0 };
			float rgba = DecodeFloatRGBA(color);
			surf2Dwrite(rgba, raycastingSurface, 4 * pixel.x, pixel.y);
		}*/
			
	}
}

