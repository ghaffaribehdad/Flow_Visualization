#include "hip/hip_runtime.h"

#include "VolumeTexture.h"
#include <string>







bool VolumeTexture3D::initialize
(
	const int3 & dimension,
	bool normalizedCoords,
	hipTextureAddressMode addressMode_x,
	hipTextureAddressMode addressMode_y,
	hipTextureAddressMode addressMode_z,
	hipTextureFilterMode _cudaTextureFilterMode
)
{

	hipExtent extent = make_hipExtent(dimension.x, dimension.y, dimension.z);

	// Allocate 3D Array
	hipChannelFormatDesc channelFormatDesc = hipCreateChannelDesc<float4>();
	hipMalloc3DArray(&this->cuArray_velocity, &channelFormatDesc, extent);



	// set copy parameters to copy from velocity field to array
	hipMemcpy3DParms cpyParams = { 0 };

	cpyParams.srcPtr = make_hipPitchedPtr((void*)this->h_field, extent.width * sizeof(float4), extent.width, extent.height);
	cpyParams.dstArray = this->cuArray_velocity;
	cpyParams.kind = hipMemcpyHostToDevice;
	cpyParams.extent = extent;

	// Copy velocities to 3D Array
	gpuErrchk(hipMemcpy3D(&cpyParams));
	// might need sync before release the host memory


	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description
	texDesc.filterMode = _cudaTextureFilterMode;
	texDesc.normalizedCoords = normalizedCoords;

	texDesc.addressMode[0] = addressMode_x;
	texDesc.addressMode[1] = addressMode_y;
	texDesc.addressMode[2] = addressMode_z;
	texDesc.readMode = hipReadModeElementType;

	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return true;

}



bool VolumeTexture3D::initialize_devicePointer
(
	const int3 & dimension,
	bool normalizedCoords,
	hipTextureAddressMode addressMode_x,
	hipTextureAddressMode addressMode_y,
	hipTextureAddressMode addressMode_z,
	hipTextureFilterMode _cudaTextureFilterMode
)
{

	hipExtent extent = make_hipExtent(dimension.x, dimension.y, dimension.z);

	// Allocate 3D Array
	hipChannelFormatDesc channelFormatDesc = hipCreateChannelDesc<float4>();
	gpuErrchk(hipMalloc3DArray(&this->cuArray_velocity, &channelFormatDesc, extent));



	// set copy parameters to copy from velocity field to array
	hipMemcpy3DParms cpyParams = { 0 };

	cpyParams.srcPtr = make_hipPitchedPtr((void*)this->h_field, extent.width * sizeof(float4), extent.width, extent.height);
	cpyParams.dstArray = this->cuArray_velocity;
	cpyParams.kind = hipMemcpyDeviceToDevice;
	cpyParams.extent = extent;

	// Copy velocities to 3D Array
	gpuErrchk(hipMemcpy3D(&cpyParams));
	// might need sync before release the host memory


	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description
	texDesc.filterMode = _cudaTextureFilterMode;
	texDesc.normalizedCoords = normalizedCoords;

	texDesc.addressMode[0] = addressMode_x;
	texDesc.addressMode[1] = addressMode_y;
	texDesc.addressMode[2] = addressMode_z;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));



	return true;

}






bool VolumeTexture3D::initialize_array
(
	bool normalizedCoords,
	hipTextureAddressMode addressMode_x,
	hipTextureAddressMode addressMode_y,
	hipTextureAddressMode addressMode_z,
	hipTextureFilterMode _cudaTextureFilterMode
)
{

	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description
	texDesc.normalizedCoords = normalizedCoords;
	texDesc.filterMode = _cudaTextureFilterMode;
	texDesc.addressMode[0] = addressMode_x;
	texDesc.addressMode[1] = addressMode_y;
	texDesc.addressMode[2] = addressMode_z;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return true;

}


void VolumeTexture3D::release()
{
	gpuErrchk(hipDestroyTextureObject(this->t_field));
	gpuErrchk(hipFreeArray(this->cuArray_velocity));

}





bool VolumeTexture2D::initialize
(
	const int2 & gridSize,
	bool normalizedCoords,
	hipTextureAddressMode addressMode_x,
	hipTextureAddressMode addressMode_y,
	hipTextureFilterMode _cudaTextureFilterMode
)
{


	// Allocate 2D Array
	hipChannelFormatDesc channelFormatDesc = hipCreateChannelDesc<float4>();
	gpuErrchk(hipMallocArray(&cuArray_velocity, &channelFormatDesc, gridSize.x, gridSize.y));


	gpuErrchk(hipMemcpy2DToArray(this->cuArray_velocity,0,0,h_field, gridSize.x * sizeof(float4), gridSize.x *sizeof(float4), gridSize.y,hipMemcpyHostToDevice));



	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description
	texDesc.normalizedCoords = normalizedCoords;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.addressMode[0] = addressMode_x;
	texDesc.addressMode[1] = addressMode_y;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return true;

}


bool VolumeTexture2D::initialize_array
(
	bool normalizedCoords,
	hipTextureAddressMode addressMode_x,
	hipTextureAddressMode addressMode_y,
	hipTextureFilterMode _cudaTextureFilterMode

)
{

	
	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description
	texDesc.normalizedCoords = normalizedCoords;
	texDesc.filterMode = _cudaTextureFilterMode;
	texDesc.addressMode[0] = addressMode_x;
	texDesc.addressMode[1] = addressMode_y;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return true;

}


void VolumeTexture2D::release()
{

	hipFreeArray(this->cuArray_velocity);
	hipDestroyTextureObject(this->t_field);

}






bool VolumeTexture1D::initialize
(
	size_t width,
	bool normalizedCoords,
	hipTextureAddressMode addressMode_x,
	hipTextureFilterMode _cudaTextureFilterMode

)
{

	// Allocate 2D Array
	hipChannelFormatDesc channelFormatDesc = hipCreateChannelDesc<float>();
	gpuErrchk(hipMallocArray(&cuArray_velocity, &channelFormatDesc, width));


	gpuErrchk(hipMemcpy2DToArray(this->cuArray_velocity, 0, 0, h_field, width * sizeof(float), width * sizeof(float), 0, hipMemcpyHostToDevice));

	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description

	texDesc.filterMode = _cudaTextureFilterMode;
	texDesc.normalizedCoords = normalizedCoords;
	texDesc.addressMode[0] = addressMode_x;
	texDesc.readMode = hipReadModeElementType;

	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return true;

}


bool VolumeTexture1D::initialize_array
(

	bool normalizedCoords,
	hipTextureAddressMode addressMode_x,
	hipTextureFilterMode _cudaTextureFilterMode

)
{

	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description

	texDesc.filterMode = _cudaTextureFilterMode;
	texDesc.normalizedCoords = normalizedCoords;
	texDesc.addressMode[0] = addressMode_x;
	texDesc.readMode = hipReadModeElementType;

	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return true;

}



void VolumeTexture1D::release()
{

	hipFreeArray(this->cuArray_velocity);
	hipDestroyTextureObject(this->t_field);

}


// Explicit declaration
template VolumeTexture3D_T<float4>;
template VolumeTexture3D_T<float3>;
template VolumeTexture3D_T<float>;

template<typename T>
bool VolumeTexture3D_T<T>::initialize
(
	const int3 & dimension,
	bool normalizedCoords,
	hipTextureAddressMode addressMode_x,
	hipTextureAddressMode addressMode_y,
	hipTextureAddressMode addressMode_z,
	hipTextureFilterMode _cudaTextureFilterMode
)
{

	hipExtent extent = make_hipExtent(dimension.x, dimension.y, dimension.z);

	// Allocate 3D Array
	hipChannelFormatDesc channelFormatDesc = hipCreateChannelDesc<float4>();
	hipMalloc3DArray(&this->cuArray_velocity, &channelFormatDesc, extent);



	// set copy parameters to copy from velocity field to array
	hipMemcpy3DParms cpyParams = { 0 };

	cpyParams.srcPtr = make_hipPitchedPtr((void*)this->h_field, extent.width * sizeof(T), extent.width, extent.height);
	cpyParams.dstArray = this->cuArray_velocity;
	cpyParams.kind = hipMemcpyHostToDevice;
	cpyParams.extent = extent;

	// Copy velocities to 3D Array
	gpuErrchk(hipMemcpy3D(&cpyParams));
	// might need sync before release the host memory


	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description
	texDesc.filterMode = _cudaTextureFilterMode;
	texDesc.normalizedCoords = normalizedCoords;

	texDesc.addressMode[0] = addressMode_x;
	texDesc.addressMode[1] = addressMode_y;
	texDesc.addressMode[2] = addressMode_z;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return true;

}


template<typename T>
bool VolumeTexture3D_T<T>::initialize_array
(
	bool normalizedCoords,
	hipTextureAddressMode addressMode_x,
	hipTextureAddressMode addressMode_y,
	hipTextureAddressMode addressMode_z,
	hipTextureFilterMode _cudaTextureFilterMode
)
{

	// Set Texture Description
	hipTextureDesc texDesc;
	hipResourceDesc resDesc;
	hipResourceViewDesc resViewDesc;

	memset(&resDesc, 0, sizeof(resDesc));
	memset(&texDesc, 0, sizeof(texDesc));
	memset(&resViewDesc, 0, sizeof(resViewDesc));



	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = this->cuArray_velocity;

	// Texture Description
	texDesc.normalizedCoords = normalizedCoords;
	texDesc.filterMode = _cudaTextureFilterMode;
	texDesc.addressMode[0] = addressMode_x;
	texDesc.addressMode[1] = addressMode_y;
	texDesc.addressMode[2] = addressMode_z;
	texDesc.readMode = hipReadModeElementType;



	// Create the texture and bind it to the array
	gpuErrchk(hipCreateTextureObject(&this->t_field, &resDesc, &texDesc, NULL));

	return true;

}

template<typename T>
void VolumeTexture3D_T<T>::release()
{
	hipFreeArray(this->cuArray_velocity);
	hipDestroyTextureObject(this->t_field);

}
