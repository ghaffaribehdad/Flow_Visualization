#include "hip/hip_runtime.h"
#include "ftleHelperFunctions.h"
#include "../Cuda/hip/hip_vector_types.h"
#include "../Particle/Particle.h"
#include "../Cuda/CudaHelperFunctions.h"

__global__ void  traceDispersion3D_path_FTLE
(
	Particle* particle,
	hipSurfaceObject_t heightFieldSurface3D,
	hipSurfaceObject_t heightFieldSurface3D_extra,
	hipTextureObject_t velocityField_0,
	hipTextureObject_t velocityField_1,
	SolverOptions solverOptions,
	DispersionOptions dispersionOptions,
	RK4STEP RK4step,
	int timestep
) 
{
	// Extract dispersion options
	int nParticles = dispersionOptions.gridSize_2D[0] * dispersionOptions.gridSize_2D[1];

	int index = CUDA_INDEX;

	if (index < nParticles)
	{
		float3 gridDiameter = make_float3(solverOptions.gridDiameter[0], solverOptions.gridDiameter[1], solverOptions.gridDiameter[2]);



		// find the index of the particle (!!!!must be revised!!!!)
		int index_y = index / dispersionOptions.gridSize_2D[1];
		int index_x = index - (index_y * dispersionOptions.gridSize_2D[1]);


		// Trace particle using RK4 

		switch (RK4step)
		{
		case RK4STEP::ODD:
			for (int i = 0; i < FTLE_NEIGHBOR; i++)
			{
				RK4Path(velocityField_0, velocityField_1, &particle[index * FTLE_NEIGHBOR + i], gridDiameter, dispersionOptions.dt, true);
			}
			break;

		case RK4STEP::EVEN:
			for (int i = 0; i < 7; i++)
			{
				RK4Path(velocityField_1, velocityField_0, &particle[index * FTLE_NEIGHBOR + i], gridDiameter, dispersionOptions.dt, true);
			}
			break;
		}

		float ftle = FTLE3D(&particle[index * FTLE_NEIGHBOR], dispersionOptions.ftleDistance, dispersionOptions.dt * (timestep+1));

		// extract the height
		float3 position = particle[index * FTLE_NEIGHBOR].m_position;
		float3 velocity = particle[index * FTLE_NEIGHBOR].m_velocity;

		float4 heightTexel = { position.y,0.0,0.0,ftle };
		float4 extraTexel = { 1.0f, 0.0f ,0.0f, 0.0f};

		
		// copy it in the surface3D
		surf3Dwrite(heightTexel, heightFieldSurface3D, sizeof(float4) * index_x, index_y, timestep);
		surf3Dwrite(extraTexel, heightFieldSurface3D_extra, sizeof(float4) * index_x, index_y, timestep);

	}
}



__device__ float FTLE3D(Particle* particles, float distance, float T)
{
	fMat3X3 d_Flowmap(0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f);
	
	// Calculate the Jacobian of the Flow Map
	d_Flowmap.r1.x = (particles[1].m_position.x - particles[2].m_position.x) / distance;
	d_Flowmap.r1.y = (particles[3].m_position.x - particles[4].m_position.x) / distance;
	d_Flowmap.r1.z = (particles[5].m_position.x - particles[6].m_position.x) / distance;

	d_Flowmap.r2.x = (particles[1].m_position.y - particles[2].m_position.y) / distance;
	d_Flowmap.r2.y = (particles[3].m_position.y - particles[4].m_position.y) / distance;
	d_Flowmap.r2.z = (particles[5].m_position.y - particles[6].m_position.y) / distance;

	d_Flowmap.r3.x = (particles[1].m_position.z - particles[2].m_position.z) / distance;
	d_Flowmap.r3.y = (particles[3].m_position.z - particles[4].m_position.z) / distance;
	d_Flowmap.r3.z = (particles[5].m_position.z - particles[6].m_position.z) / distance;

	// Find the Delta Tensor
	fMat3X3 td_Flowmap = transpose(d_Flowmap);
	fMat3X3 delta = mult(td_Flowmap, d_Flowmap);
	float3 eigen = { 0.0f,0.0f,0.0f };
	
	// Calculate and sort the eigenvalues
	eigensolveHasan(delta, eigen);

	float lambda_max = eigen.z;


	return (1.0f/T) * logf(sqrtf(lambda_max));
}