#include "Pathlinesolver.h"
#include "CudaHelperFunctions.h"



bool PathlineSolver::release()
{
	this->volume_IO.release();
	hipFree(this->d_Particles);


	this->volumeTexture_0.release();
	this->volumeTexture_1.release();

	return true;
}



__host__ bool PathlineSolver::solve()
{
	//At least two timesteps is needed
	int timeSteps = solverOptions->lastIdx - solverOptions->currentIdx;

	// Initialize Volume IO (Save file path and file names)
	this->volume_IO.Initialize(this->solverOptions);

	// Initialize Particles and upload it to GPU
	this->InitializeParticles(solverOptions->seedingPattern);

	int blockDim = 256;
	int thread = (this->solverOptions->lines_count / blockDim) + 1;
	
	solverOptions->lineLength = timeSteps;
	bool odd = false;

	// set solverOptions once


	// we go through each time step and solve RK4 for even time steps the first texture is updated,
	// while the second texture is updated for odd time steps



	for (int step = 0; step < timeSteps; step++)
	{
		// First Step

		switch (solverOptions->Compressed)
		{

			case true: // Compressed Data
			{
				if (step == 0)
				{
					loadTextureCompressed(solverOptions, volumeTexture_0, solverOptions->currentIdx);
					loadTextureCompressed(solverOptions, volumeTexture_1, solverOptions->currentIdx + 1);
				}
				else if (step % 2 == 0) // => EVEN
				{
					this->volumeTexture_1.release();
					loadTextureCompressed(solverOptions, volumeTexture_1, solverOptions->currentIdx + step + 1);
					odd = false;
				}
				else if (step % 2 != 0) // => ODD
				{
					this->volumeTexture_0.release();
					loadTextureCompressed(solverOptions, volumeTexture_0, solverOptions->currentIdx + step + 1);
					odd = true;

				}

				break;
			}

			case false: // Uncompressed Data
			{
				if (step == 0)
				{
					loadTexture(solverOptions, volumeTexture_0, solverOptions->currentIdx);
					loadTexture(solverOptions, volumeTexture_1, solverOptions->currentIdx + 1);
				}
				else if (step % 2 == 0) // => EVEN
				{
					this->volumeTexture_1.release();
					loadTexture(solverOptions, volumeTexture_1, solverOptions->currentIdx + step + 1);
					odd = false;

				}
				else if (step % 2 != 0) // => ODD
				{
					this->volumeTexture_0.release();
					loadTexture(solverOptions, volumeTexture_0, solverOptions->currentIdx + step + 1);
					odd = true;
				}
				break;
			}

		}
	
		//int numofBlock = 0;
		//hipOccupancyMaxActiveBlocksPerMultiprocessor(&numofBlock, TracingPath, blockDim, 0);
		//std::printf("Optimized number of blocks are  %d \n", numofBlock);	

		TracingPath << <blockDim, thread >> > (this->d_Particles, volumeTexture_0.getTexture(), volumeTexture_1.getTexture(), *solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer), odd, step);
		std::printf("\n\n");
	}  	


	this->release();
	return true;
}

__host__ bool PathlineSolver::initializeRealtime(SolverOptions * p_solverOptions)
{

	this->solverOptions = p_solverOptions;
	this->InitializeCUDA();
	this->volume_IO.Initialize(p_solverOptions);
	this->InitializeParticles(this->solverOptions->seedingPattern);
	
	return true;
}


__host__ bool PathlineSolver::resetRealtime()
{

	this->release();
	
	return true;
}

__host__ bool PathlineSolver::solveRealtime(int & pathCounter)
{
	int blockDim = 256;
	int thread = (this->solverOptions->lines_count / blockDim) + 1;
	
	bool odd = false;


	// First Step

	switch (solverOptions->Compressed)
	{

	case true: // Compressed Data
	{
		if (pathCounter == 0)
		{
			loadTextureCompressed(solverOptions, volumeTexture_0, solverOptions->firstIdx);
			loadTextureCompressed(solverOptions, volumeTexture_1, solverOptions->firstIdx + 1);
		}
		else if (pathCounter % 2 == 0) // => EVEN
		{
			this->volumeTexture_1.release();
			loadTextureCompressed(solverOptions, volumeTexture_1, solverOptions->firstIdx + pathCounter + 1);
			odd = false;
		}
		else if (pathCounter % 2 != 0) // => ODD
		{
			this->volumeTexture_0.release();
			loadTextureCompressed(solverOptions, volumeTexture_0, solverOptions->firstIdx + pathCounter + 1);
			odd = true;

		}

		break;
	}

	case false: // Uncompressed Data
	{
		if (pathCounter == 0)
		{
			loadTexture(solverOptions, volumeTexture_0, solverOptions->firstIdx);
			loadTexture(solverOptions, volumeTexture_1, solverOptions->firstIdx + 1);
		}
		else if (pathCounter % 2 == 0) // => EVEN
		{
			this->volumeTexture_1.release();
			loadTexture(solverOptions, volumeTexture_1, solverOptions->firstIdx + pathCounter + 1);
			odd = false;

		}
		else if (pathCounter % 2 != 0) // => ODD
		{
			this->volumeTexture_0.release();
			loadTexture(solverOptions, volumeTexture_0, solverOptions->firstIdx + pathCounter + 1);
			odd = true;
		}
		break;
	}

	}

	if (pathCounter == solverOptions->lineLength)
	{
		resetRealtime();
		pathCounter = 0;

	}
	else
	{
		TracingPath << <blockDim, thread >> > (this->d_Particles, volumeTexture_0.getTexture(), volumeTexture_1.getTexture(), *solverOptions, reinterpret_cast<Vertex*>(this->p_VertexBuffer), odd, pathCounter);
		std::printf("\n\n");
		pathCounter++;
	}
	
	solverOptions->counter = pathCounter;
	
	return true;
}


